#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //lib W10
#include ""//lib W10

#include <iostream> // prompt Output
#include <fstream> //File Output
#include <math.h> //lib mayh
#include <stdio.h> // lib stantard
#include <hip/hip_fp16.h> // lib CUDA
#include <windows.h>
#include <fstream>
#include <string>


#define Dim_isFix 0;
#define Dim_end 20.0f;
#define Dim_start -20.0f;
#define Dim_NbStep 4;
#define Dim_step 10.0f;

#define ITER_MAX 255;
#define ITER_MIN 1;
#define ITER_isFix 0;

#define DEV 1;
#define FILTER 0;
#define POWER 2.0f;
#define ISSHOW 1;
#define RMAX 4.0f;

#define NBPOINTS 64;

#define MODEAUTOMANU false;
//Strcture state

 struct struct_Stat_float_T {
	float Xmin = Dim_end
	float Xmax = Dim_start
	float Wmin = Dim_end
	float Wmax = Dim_start
	float Ymin = Dim_end
	float Ymax = Dim_start
	float Zmin = Dim_end
	float Zmax = Dim_start
	float Wstep = 0.0f;
	float Xstep = 0.0f;
	float Ystep = 0.0f;
	float Zstep = 0.0f;
	unsigned long NbPoint = 0;
 };


// struct sur la gestion des dimensions
 struct struct_P_float_T {
	 int isFix = Dim_isFix
	 float start = Dim_start
	 float end = Dim_end
	 int NbStep = Dim_NbStep
	 float step = Dim_step
		int coef = 1;
} ;


struct 	 struct_Iter_T  {
	int max = ITER_MAX
	int min = ITER_MIN
	int isFix = ITER_isFix
} ;

typedef struct 	struct_FileName_T {
	char root[100];
	char txt[110];
	char csv[110];
	char stat[110];
	char histo[110];
	char stl[110];
} struct_FileName_T;

// struct sur la gestion param�tres d'entr�
struct struct_P_All_T {
	struct_P_float_T X;
	struct_P_float_T Y;
	struct_P_float_T Z;
	struct_P_float_T W;
	struct_Iter_T Iter;
	int dev = DEV
		int filter = FILTER
		float power = POWER
		int isShow = ISSHOW
		struct_FileName_T nameFile;
	float rMax = RMAX
		int NbPointByStep = NBPOINTS
		bool modeAM = MODEAUTOMANU
} ;

struct 	struct_P_Simulation_T {
	//Quaternions
	struct_P_float_T X;
	struct_P_float_T Y;
	struct_P_float_T Z;
	struct_P_float_T W;
	struct_Iter_T Iter;
	float rMax;
	//Parametrer variable systematique
	float power;
	int max;
} ;

struct struct_Q_T {
	float x;
	float y;
	float z;
	float w;
} ;

struct struct_QD_T {
	double x;
	double y;
	double z;
	double w;
};


__device__  void CreateQ_By_float(struct_QD_T *out, double x, double y, double z, double w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

//__device__  float  Get_QNorm(struct_Q_T *Q)
//{
//	float R2 = Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w;
//	float N = norm4df(Q->x, Q->y, Q->z, Q->w);
//	float R = sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
//	return R;
//}

__device__ void Get_QPow(struct_QD_T *Q, float power)
{
	
	double A = norm4d(Q->x, Q->y, Q->z, Q->w);
	//printf("A=%f 1er\n", A);
	double theta = 0.0;
	double B = 0.0;
	double R = 0.0;
	double Temp = 0.0;
	double Temp2 = 0.0;
	if (power > 0.0f && A>0.00001)
	{
		//double coef = 1.0;
		//if (A<1.0)
		//{
		//	//printf("1.0f > %f(A) *******\n", A);
		//	coef = 1 / A;
		//	Q->x = __dmul_rn(Q->x, coef);
		//	Q->y = __dmul_rn(Q->y, coef);
		//	Q->z = __dmul_rn(Q->z, coef);
		//	Q->w = __dmul_rn(Q->w, coef);
		//	if (Q->x == NAN || Q->y == NAN || Q->z == NAN || Q->w == NAN)
		//		return;
		//	//printf("coef=%f\n", coef);
		//	A = 1.0;
		//	//printf("A=%f=%f 2eme\n", A, norm4df(Q->x, Q->y, Q->z, Q->w));

		//}
		
		
		/*printf("Q->w / A = %f/%f \n",  Q->w, A);
		printf("Q->w / A = %f \n", Q->w / A, Q->w,A);*/

		//printf("%f +++++++++\n", A);
		Temp = __ddiv_rd(Q->w, A);
		if (abs(Temp) <= 1.0)
		{
			Temp2 = acos(Temp);
			theta = __dmul_rn(Temp2, power);
		}
		else
		{
			printf("Error __ddiv_rd(Q->w, A)=__ddiv_rd(%f, %f)=%f", Q->w, A, Temp);
		}
		
		if (abs(theta) <= 0.001)
			theta = 0.0;

		//B = __dmul_rn(A,A) - __dmul_rn(Q->w,Q->w);
		B = norm3d(Q->x, Q->y, Q->z);
		//caluer debug

	/*	R = pow(A / coef, (double)(power));
		if (R == NAN )
			return;*/
		double C = log(A); //__ddiv_rn(A , coef)
		C = __dmul_rn( C,power);
		if (C > -2.661474648)
			R = exp(C);
		else
		{
			Q->w = 0.0;
			Q->x = 0.0;
			Q->y = 0.0;
			Q->z = 0.0;
			return;
			//printf("%f +++++++++\n", R);
		}
			
		//R = powf(A / coef, pow);
		Temp = sin(theta);
		if (abs(Temp) > 0.001)
		{
			Temp2 = __dmul_rn(R, Temp);
			if (abs(B) > 0.01 || abs(Q->x) > 0.01)
				Q->x = __dmul_rn(Temp2, (Q->x / B));
			else
				Q->x = 0;

			Temp2 = __dmul_rn(R, Temp);
			if (abs(B) > 0.01 || abs(Q->y) > 0.01)
				Q->y = __dmul_rn(Temp2, (Q->y / B));
			else
				Q->y = 0;

			Temp2 = __dmul_rn(R, Temp);
			if (abs(B) > 0.01 || abs(Q->z) > 0.01)
				Q->z = __dmul_rn(Temp2, (Q->z / B));
			else
				Q->z = 0;
		}
		else
		{
			Q->x = 0.0;
			Q->y = 0.0;
			Q->z = 0.0;

		}


		Temp = cos(theta);
		if (abs(Temp) > 0.001)
			Q->w = __dmul_rn(R, Temp);
		else
			Q->w = 0.0;


	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0;
		Q->x = 0.0;
		Q->y = 0.0;
		Q->z = 0.0;
	}
}


__host__  void CreateQ_By_float_H(struct_Q_T *out, float x, float y, float z, float w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

__host__  float  Get_QNorm_H(struct_Q_T *Q)
{
	return sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
}

__host__ void Get_QPow_H(struct_Q_T *Q, float pow)
{
	float A = Get_QNorm_H(Q);
	float theta = 0.0f;
	float B = 0.0f;
	float R = 0.0f;
	if (pow > 0.0f && A>0.000001f)
	{
		float coef = 1.0f;
		if (A<1.0f)
		{
			//printf("%f *******\n", A);
			coef = 1 / A;
			Q->x *= coef;
			Q->y *= coef;
			Q->z *= coef;
			Q->w *= coef;

		}
		A = Get_QNorm_H(Q);
		//printf("%f +++++++++\n", A);
		theta = acosf(Q->w / A)*pow;
		B = sqrt(A*A - Q->w*Q->w);
		R = exp2f(logf(A / coef)* pow);
		Q->x = R*sinf(theta)*(Q->x / B);
		Q->y = R*sinf(theta)*(Q->y / B);
		Q->z = R*sinf(theta)*(Q->z / B);
		Q->w = R*cosf(theta);

	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0f;
		Q->x = 0.0f;
		Q->y = 0.0f;
		Q->z = 0.0f;

	}
}

__host__ int  GetQIter_H(struct_P_Simulation_T *P_Simulation_DEVICE, int  *x_filter, int  *y_filter, int *z_filter, int *w_filter)
{
	//int Tempindex = 0;
	struct_Q_T Q_Current;
	float w, x, y, z;
	int iter_computed;
	//X
	x = ((float)*x_filter)*P_Simulation_DEVICE->X.step + P_Simulation_DEVICE->X.start;
	//Y
	y = ((float)*y_filter)*P_Simulation_DEVICE->Y.step + P_Simulation_DEVICE->Y.start;
	//Z
	z = ((float)*z_filter)*P_Simulation_DEVICE->Z.step + P_Simulation_DEVICE->Z.start;
	//W
	w = ((float)*w_filter)*P_Simulation_DEVICE->W.step + P_Simulation_DEVICE->W.start;



	CreateQ_By_float_H(&Q_Current, x, y, z, w);

	for (iter_computed = 0; iter_computed <= P_Simulation_DEVICE->Iter.max; iter_computed++)
	{
		Get_QPow_H(&Q_Current, P_Simulation_DEVICE->power);
		Q_Current.x += x;
		Q_Current.y += y;
		Q_Current.z += z;
		Q_Current.w += w;

		if (Get_QNorm_H(&Q_Current) > P_Simulation_DEVICE->rMax)
		{
			if (iter_computed > 0)
				iter_computed--;
			return iter_computed;
		}
	}
	if (iter_computed > 0)
		iter_computed--;
	return iter_computed;
}
__host__ bool  FilterQ_H(int *Filter, int *Nx, int *Ny, int *Nz, int *Nw, int iter,	struct_P_Simulation_T *P_Simulation)
{
	if (*Filter == 0)
		return true;
	int iter_computed = 0;

	int pasx = 1;
	if (P_Simulation->X.NbStep == 1)
		pasx = 0;

	int pasy = 1;
	if (P_Simulation->Y.NbStep == 1)
		pasy = 0;

	int pasz = 1;
	if (P_Simulation->Z.NbStep == 1)
		pasz = 0;

	int pasw = 1;
	if (P_Simulation->W.NbStep == 1)
		pasw = 0;


	for (int x_filter = *Nx - pasx; x_filter <= *Nx + pasx; x_filter++)
	{
		for (int y_filter = *Ny - pasy; y_filter <= *Ny + pasy; y_filter++)
		{
			for (int z_filter = *Nz - pasz; z_filter <= *Nz + pasz; z_filter++)
			{
				for (int w_filter = *Nw - pasw; w_filter <= *Nw + pasw; w_filter++)
				{
					iter_computed = GetQIter_H(P_Simulation, &x_filter, &y_filter, &z_filter, &w_filter);
					if (*Filter == 1)
					{
						if (iter_computed != iter)
							return true;
					}
					else //filter==2
					{
						if (iter_computed == 0)
							return true;
					}
				}
			}
		}
	}
	return false;
}


int menu(int argc, char *argv[], struct_P_All_T *Config)
{
	//Config

	strcpy(Config->nameFile.root, "O");
	if (argc == 1)
	{
		Config->W.end = -0.3375f;
		Config->W.start = -0.3375f;
		Config->W.step = 1.0f;
		Config->W.isFix = 1;
		Config->W.NbStep = 0;

		float borne = -0.75f;
		Config->X.isFix = 2;
		Config->X.start = borne;
		Config->X.end = 0.0f;
		Config->X.NbStep = 1;

		Config->Y.isFix = 2;
		Config->Y.start = borne;
		Config->Y.end = 0.0f;
		Config->Y.NbStep = 1;

		Config->Z.isFix = 2;
		Config->Z.start = borne;
		Config->Z.end = 0.0f;
		Config->Z.NbStep = 1;

		/*Config->Z.end = -0.0;
		Config->Z.start = -0.0;
		Config->Z.step = 1.0;
		Config->Z.isFix = 1;
		Config->Z.NbStep = 1;*/



		Config->filter = 2;


		Config->modeAM = false;
		//Config->Iter.isFix = 1;
		//Config->Iter.max = 2;
		Config->Iter.min = 10;
		//Config->rMax = 2;
	}




	//Arg Help
	char Str_H[] = "-h";
	char Str_Help[] = "--help";

	//Arg X
	char Str_xFix[] = "-x";
	char Str_xMax[] = "-xmax";
	char Str_xMin[] = "-xmin";
	char Str_xNbStep[] = "-xNbStep";
	//Arg W
	char Str_wFix[] = "-w";
	char Str_wMax[] = "-wmax";
	char Str_wMin[] = "-wmin";
	char Str_wNbStep[] = "-wNbStep";
	//Arg Y
	char Str_yFix[] = "-y";
	char Str_yMax[] = "-ymax";
	char Str_yMin[] = "-ymin";
	char Str_yNbStep[] = "-yNbStep";
	//Arg Z
	char Str_zFix[] = "-z";
	char Str_zMax[] = "-zmax";
	char Str_zMin[] = "-zmin";
	char Str_zNbStep[] = "-zNbStep";
	//Arg Iter
	char Str_IterFix[] = "-iter";
	char Str_IterMax[] = "-iterMax";
	char Str_IterMin[] = "-iterMin";
	//Arg Dev
	char Str_dev[] = "-device";
	//Arg Filter
	char Str_filter[] = "-filter";
	//Arg Power
	char Str_power[] = "-power";
	//Arg IsShow
	char Str_IsShow[] = "-isShow";
	//Arg Output File
	char Str_Out[] = "-o";
	//Arg Rmax
	char Str_rMax[] = "-rMax";

	//Arg Rmax
	char Str_mode[] = "-mode";
	char Str_modeAuto[] = "auto";
	char Str_modeManu[] = "manu";
	//Si il y a des Arguments
	if (argc > 1)
	{
		// dédection la commande -help pu -h
		for (int i = 1; i < argc; i++)
		{
			if (strcmp(argv[i], Str_Help) == 0 || strcmp(argv[i], Str_H) == 0)
			{
				//Affiche Help
				FILE *fileman;
				char line[1000];
				fileman = fopen("man", "r");
				if (fileman != NULL)
				{
					while (std::fgets(line, 1000, fileman))
					{
						std::cout << line;
					}
					fclose(fileman);
					return 0; //Fin du programme
				}
				else
				{
					std::cout << "Error  : man no found" << "\n";
					return -1;
				}
			}
		}
		//Verification : si nb d'arguments est paire --> erreur
		if (argc % 2 == 0)
		{
			std::cout << "Error 00 : Argument impaire" << "\n";
			return -1;
		}
		else //: si nb d'arguments est impaire --> fonctionement normale
		{
			for (int i = 1; i < argc; i += 2)
			{
				std::cout << "Analyse du couple d'arguments :  " << argv[i] << " " << argv[i + 1] << "\n";
				if (strcmp(argv[i], Str_xFix) == 0)
				{
					if (Config->X.isFix == 0) //Si premier config
					{
						Config->X.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xFix << ": value is not float " << "\n";
							return -1;
						}
						Config->X.start = value;
						Config->X.end = value;
						Config->X.NbStep = 0;
						Config->X.step = 0.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_xFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wFix) == 0)
				{
					if (Config->W.isFix == 0) //Si premier config
					{
						Config->W.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wFix << ": value is not float " << "\n";
							return -1;
						}
						Config->W.start = value;
						Config->W.end = value;
						Config->W.NbStep = 0;
						Config->W.step = 0.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_wFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yFix) == 0)
				{
					if (Config->Y.isFix == 0) //Si premier config
					{
						Config->Y.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yFix << ": value is not float " << "\n";
							return -1;
						}
						Config->Y.start = value;
						Config->Y.end = value;
						Config->Y.NbStep = 0;
						Config->Y.step = 0.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_yFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zFix) == 0)
				{
					if (Config->Z.isFix == 0) //Si premier config
					{
						Config->Z.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zFix << ": value is not float " << "\n";
							return -1;
						}
						Config->Z.start = value;
						Config->Z.end = value;
						Config->Z.NbStep = 0;
						Config->Z.step = 0.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_zFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xMax) == 0)
				{
					if (Config->X.isFix == 0 || Config->X.isFix == 2)
					{
						Config->X.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xMax << ": value is not float " << "\n";
							return -1;
						}
						Config->X.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wMax) == 0)
				{
					if (Config->W.isFix == 0 || Config->W.isFix == 2)
					{
						Config->W.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wMax << ": value is not float " << "\n";
							return -1;
						}
						Config->W.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yMax) == 0)
				{
					if (Config->Y.isFix == 0 || Config->Y.isFix == 2)
					{
						Config->Y.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yMax << ": value is not float " << "\n";
							return -1;
						}
						Config->Y.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zMax) == 0)
				{
					if (Config->Z.isFix == 0 || Config->Z.isFix == 2)
					{
						Config->Z.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zMax << ": value is not float " << "\n";
							return -1;
						}
						Config->Z.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_zMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xMin) == 0)
				{
					if (Config->X.isFix == 0 || Config->X.isFix == 2)
					{
						Config->X.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xMin << ": value is not float " << "\n";
							return -1;
						}
						Config->X.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wMin) == 0)
				{
					if (Config->W.isFix == 0 || Config->W.isFix == 2)
					{
						Config->W.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wMin << ": value is not float " << "\n";
							return -1;
						}
						Config->W.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yMin) == 0)
				{
					if (Config->Y.isFix == 0 || Config->Y.isFix == 2)
					{
						Config->Y.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yMin << ": value is not float " << "\n";
							return -1;
						}
						Config->Y.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zMin) == 0)
				{
					if (Config->Z.isFix == 0 || Config->Z.isFix == 2)
					{
						Config->Z.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zMin << ": value is not float " << "\n";
							return -1;
						}
						Config->Z.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_zMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xNbStep) == 0)
				{
					if (Config->X.isFix == 0 || Config->X.isFix == 2)
					{
						Config->X.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_xNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config->X.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wNbStep) == 0)
				{
					if (Config->W.isFix == 0 || Config->W.isFix == 2)
					{
						Config->W.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_wNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config->W.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yNbStep) == 0)
				{
					if (Config->Y.isFix == 0 || Config->Y.isFix == 2)
					{
						Config->Y.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_yNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config->Y.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zNbStep) == 0)
				{
					if (Config->Z.isFix == 0 || Config->Z.isFix == 2)
					{
						Config->Z.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_yNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config->Z.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterFix) == 0)
				{
					if (Config->Iter.isFix == 0) //Si premier config
					{
						Config->Iter.isFix = 1;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_IterFix << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config->Iter.max = value;
						Config->Iter.min = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterMax) == 0)
				{
					if (Config->Iter.isFix == 0 || Config->Iter.isFix == 2) //Si premier config
					{
						Config->Iter.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_IterMax << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config->Iter.max = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterMin) == 0)
				{
					if (Config->Iter.isFix == 0 || Config->Iter.isFix == 2) //Si premier config
					{
						Config->Iter.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_IterMin << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config->Iter.min = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_dev) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_dev << ": value is not int" << "\n";
						return -1;
					}
					Config->dev = value;
				}
				else if (strcmp(argv[i], Str_filter) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_filter << ": value is not int" << "\n";
						return -1;
					}
					Config->filter = value;
				}
				else if (strcmp(argv[i], Str_power) == 0)
				{
					float value = (float)atof(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_filter << ": value is not float" << "\n";
						return -1;
					}
					Config->power = value;
				}
				else if (strcmp(argv[i], Str_IsShow) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_IsShow << ": value is not int" << "\n";
						return -1;
					}
					Config->isShow = value;
				}
				else if (strcmp(argv[i], Str_rMax) == 0)
				{
					float value = (float)atof(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_rMax << ": value is not float" << "\n";
						return -1;
					}
					Config->rMax = value;
				}
				else if (strcmp(argv[i], Str_Out) == 0)
				{
					if (strlen(argv[i + 1])<100)
						strcpy(Config->nameFile.root, argv[i + 1]);
					else
					{
						std::cout << "Error 04 strlen fileOutput must be inf to 100 signe \n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_mode) == 0)
				{
					if (strcmp(argv[i + 1], Str_modeAuto) == 0)
						Config->modeAM = true;
					else if (strcmp(argv[i + 1], Str_modeManu) == 0)
						Config->modeAM = false;
					else
					{
						std::cout << "Error 08 mode no found\n";
						return -1;
					}
				}
				else
				{
					std::cout << "Warning 05  Arg not know : " << argv[i] << " " << argv[i + 1] << "\n";
				}
			}
		}
	}
	bool IsErrors = false;
	// Verification W
	if (Config->W.isFix == 1 && Config->W.start == Config->W.end)
	{
		;//OK
	}
	else if (Config->W.isFix == 2 || Config->W.isFix == 0)
	{
		Config->W.isFix = 2;
		if (Config->W.start < Config->W.end && Config->W.NbStep>=1)
		{
			Config->W.step = (Config->W.end - Config->W.start) / ((float)Config->W.NbStep);
		}
		else
		{
			std::cout << "Error W  :  wmax must be sup wmin AND nbStep must be sup 1\n";
			IsErrors = true;
		}
	}
	else
	{
		std::cout << "Error W  :  Error unknow\n";
		IsErrors = true;
	}
	// Verification X
	if (Config->X.isFix == 1 && Config->X.start == Config->X.end)
	{
		;//OK
	}
	else if (Config->X.isFix == 2 || Config->X.isFix == 0)
	{
		Config->X.isFix = 2;
		if (Config->X.start < Config->X.end && Config->X.NbStep>=1)
		{
			Config->X.step = (Config->X.end - Config->X.start) / ((float)Config->X.NbStep);
		}
		else
		{
			std::cout << "Error X  :  xmax must be sup xmin AND nbStep must be sup 1\n";
			IsErrors = true;
		}
	}
	else
	{
		std::cout << "Error X  :  Error unknow\n";
		IsErrors = true;
	}
	// Verification Y
	if (Config->Y.isFix == 1 && Config->Y.start == Config->Y.end)
	{
		;//OK
	}
	else if (Config->Y.isFix == 2 || Config->Y.isFix == 0)
	{
		Config->Y.isFix = 2;
		if (Config->Y.start < Config->Y.end && Config->Y.NbStep>=1)
		{
			Config->Y.step = (Config->Y.end - Config->Y.start) / ((float)Config->Y.NbStep);
		}
		else
		{
			std::cout << "Error Y  :  ymax must be sup ymin AND nbStep must be sup 1\n";
			IsErrors = true;
		}
	}
	else
	{
		std::cout << "Error Y  :  Error unknow\n";
		IsErrors = true;
	}
	// Verification Z
	if (Config->Z.isFix == 1 && Config->Z.start == Config->Z.end)
	{
		;//OK
	}
	else if (Config->Z.isFix == 2 || Config->Z.isFix == 0)
	{
		Config->Z.isFix = 2;
		if (Config->Z.start < Config->Z.end && Config->Z.NbStep>=1)
		{
			Config->Z.step = (Config->Z.end - Config->Z.start) / ((float)Config->Z.NbStep);
		}
		else
		{
			std::cout << "Error Z  :  zmax must be sup zmin AND nbStep must be sup 1\n";
			IsErrors = true;
		}
	}
	else
	{
		std::cout << "Error Z  :  Error unknow\n";
		IsErrors = true;
	}
	// Verification Iter
	if (Config->Iter.isFix == 1)
	{
		if (Config->Iter.min == Config->Iter.max && Config->Iter.max > 0)
		{
			; //OK
		}
		else
		{
			std::cout << "Error Iter  :  value must be sup at 0\n";
			IsErrors = true;
		}
	}
	else if (Config->Iter.isFix == 2 || Config->Iter.isFix == 0)
	{
		Config->Iter.isFix = 2;
		if (Config->Iter.min < Config->Iter.max)
		{
			; // OK
		}
		else
		{
			std::cout << "Error Iter  :  max must be sup min\n";
			IsErrors = true;
		}
	}
	else
	{
		std::cout << "Error Iter  :  Error unknow\n";
		IsErrors = true;
	}
	// Vérification dev
	int count;
	hipGetDeviceCount(&count);
	if (Config->dev >= 0 && Config->dev < count)
	{
		; //OK
	}
	else
	{
		std::cout << "Error dev :  dev must be between 0 and " << count - 1 << "\n";
		IsErrors = true;
	}
	//Verification filter
	if (Config->filter >= 0 && Config->filter <= 2)
	{
		if (Config->Iter.isFix == 1 && Config->filter == 2)
			Config->filter = 1;
	}
	else
	{
		std::cout << "Error filter :  filter must be between 0 and 2\n";
		IsErrors = true;
	}
	//Verification power
	if (Config->power >= 2.0f && Config->power <= 50.0f)
	{
		; //OK
	}
	else
	{
		std::cout << "Error power :  power must be between 2.0 and 50.0\n";
		IsErrors = true;
	}
	//Verification IsShow
	if (Config->isShow == 0 || Config->isShow == 1)
	{
		; //OK
	}
	else
	{
		std::cout << "Error isShow :  isShow must be between 0 and 1 \n";
		IsErrors = true;
	}
	//Verification rMax
	if (Config->rMax >0.0f)
	{
		; //OK
	}
	else
	{
		std::cout << "Error rMax :  rMax must be sup 0.0 \n";
		IsErrors = true;
	}
	if (IsErrors)
		return -1;

	// creation des fichiers :
	strcpy(Config->nameFile.csv, Config->nameFile.root);
	strcpy(Config->nameFile.histo, Config->nameFile.root);
	strcpy(Config->nameFile.stat, Config->nameFile.root);
	strcpy(Config->nameFile.stl, Config->nameFile.root);
	strcpy(Config->nameFile.txt, Config->nameFile.root);

	strcat(Config->nameFile.csv, ".csv");
	strcat(Config->nameFile.histo, ".histo");
	strcat(Config->nameFile.stat, ".stat");
	strcat(Config->nameFile.stl, ".stl");
	strcat(Config->nameFile.txt, ".txt");

	//Affichage de la config
	std::ofstream file;
	file.open(Config->nameFile.stat);
	file << "Parameters Current : " << "\n";
	file << "				W_start = " << Config->W.start << ", W_end = " << Config->W.end << ", W_Step = " << Config->W.step << ", W_NbStep = " << Config->W.NbStep << "\n";
	file << "				X_start = " << Config->X.start << ", X_end = " << Config->X.end << ", X_Step = " << Config->X.step << ", X_NbStep = " << Config->X.NbStep << "\n";
	file << "				Y_start = " << Config->Y.start << ", Y_end = " << Config->Y.end << ", Y_Step = " << Config->Y.step << ", Y_NbStep = " << Config->Y.NbStep << "\n";
	file << "				Z_start = " << Config->Z.start << ", Z_end = " << Config->Z.end << ", Z_Step = " << Config->Z.step << ", Z_NbStep = " << Config->Z.NbStep << "\n";
	file << "				Root FileOutput = " << Config->nameFile.root << "\n";
	file << "				iterMax = " << Config->Iter.max << "\n";
	file << "				iterMin = " << Config->Iter.min << "\n";
	file << "				rMax = " << Config->rMax << "\n";
	file << "				Filter = " << Config->filter << "\n";
	file << "				Power = " << Config->power << "\n";
	file << "				dev = " << Config->dev << "\n";
	file << "				IsShow = " << Config->isShow << "\n";
	file << "				NbPoints per step = " << Config->NbPointByStep << "\n";
	file << "				ouput File :  " << Config->nameFile.root << "\n";
	file << "cmd for use this configuration: " << "\n";
	file << "               " << argv[0] << "  ";
	if (Config->W.isFix == 1)
		file << Str_wFix << " " << Config->W.start << " ";
	else
	{
		file << Str_wMin << " " << Config->W.start << " ";
		file << Str_wMax << " " << Config->W.end << " ";
		file << Str_wNbStep << " " << Config->W.NbStep << " ";
	}
	if (Config->X.isFix == 1)
		file << Str_xFix << " " << Config->X.start << " ";
	else
	{
		file << Str_xMin << " " << Config->X.start << " ";
		file << Str_xMax << " " << Config->X.end << " ";
		file << Str_xNbStep << " " << Config->X.NbStep << " ";
	}
	if (Config->Y.isFix == 1)
		file << Str_yFix << " " << Config->Y.start << " ";
	else
	{
		file << Str_yMin << " " << Config->Y.start << " ";
		file << Str_yMax << " " << Config->Y.end << " ";
		file << Str_yNbStep << " " << Config->Y.NbStep << " ";
	}
	if (Config->Z.isFix == 1)
		file << Str_zFix << " " << Config->Z.start << " ";
	else
	{
		file << Str_zMin << " " << Config->Z.start << " ";
		file << Str_zMax << " " << Config->Z.end << " ";
		file << Str_zNbStep << " " << Config->Z.NbStep << " ";
	}
	if (Config->Iter.isFix == 1)
		file << Str_IterFix << " " << Config->Iter.max << " ";
	else
	{
		file << Str_IterMin << " " << Config->Iter.min << " ";
		file << Str_IterMax << " " << Config->Iter.max << " ";
	}
	file << Str_dev << " " << Config->dev << " ";
	file << Str_filter << " " << Config->filter << " ";
	file << Str_power << " " << Config->power << " ";
	file << Str_IsShow << " " << Config->isShow << " ";
	file << Str_rMax << " " << Config->rMax << " ";
	file << Str_Out << " " << Config->nameFile.root << " ";

	file << "\n";
	file.close();
	return 1;
}

__device__ void shunte(struct_QD_T *Q_Current)
{
	if (abs(Q_Current->w) <= 0.01)
		Q_Current->w = 0.0;
	if (abs(Q_Current->x) <= 0.01)
		Q_Current->x = 0.0;
	if (abs(Q_Current->y) <= 0.01)
		Q_Current->y = 0.0;
	if (abs(Q_Current->z) <= 0.01)
		Q_Current->z = 0.0;
}

// CUDA kernel to Compute itermax of quaternion
__global__ void kernel(const struct_P_Simulation_T *P_Simulation, int *Tab_Iter)
{
	//int Tempindex = 0;
	struct_QD_T Q_Current;
	double w, x, y, z;
	double N = 0.0;
	int iter = 0;
	//X
	x = __fma_rd(((double)blockIdx.x),P_Simulation->X.step , P_Simulation->X.start);
	//Y
	y = __fma_rd(((double)blockIdx.y),P_Simulation->Y.step , P_Simulation->Y.start);
	//Z
	z = __fma_rd(((double)blockIdx.z),P_Simulation->Z.step , P_Simulation->Z.start);
	//W
	w = __fma_rd(((double)threadIdx.x),P_Simulation->W.step , P_Simulation->W.start);

		CreateQ_By_float(&Q_Current, x, y, z, w);

		for (iter = 0; iter <= P_Simulation->Iter.max; iter++)
		{

			//if (Q_Current.x==NAN || Q_Current.y == NAN || Q_Current.z == NAN || Q_Current.w == NAN)
			//	goto Fin;
			shunte(&Q_Current);

			Get_QPow(&Q_Current, P_Simulation->power);

			shunte(&Q_Current);
			//if (Q_Current.x == NAN || Q_Current.y == NAN || Q_Current.z == NAN || Q_Current.w == NAN)
			//	goto Fin;
			N = Q_Current.x;
			Q_Current.x = __fadd_rn(x,N);

			N = Q_Current.y;
			Q_Current.y = __fadd_rn(y, N);

			N = Q_Current.z;
			Q_Current.z = __fadd_rn(z, N);

			N = Q_Current.w;
			Q_Current.w = __fadd_rn(w, N);

			shunte(&Q_Current);
			/*if (Q_Current.x == NAN || Q_Current.y == NAN || Q_Current.z == NAN || Q_Current.w == NAN)
			goto Fin;*/
			N = norm4d(Q_Current.x, Q_Current.y, Q_Current.z, Q_Current.w);
			//printf("%f , %f, %f, %f, %f\n", N, x, y, z, w);
			if (N == NAN)
				goto Fin;
			if (N> P_Simulation->rMax)
				goto Fin;
		}

Fin:
	if (iter > 0)
		iter--;
	
	int index = blockIdx.x*P_Simulation->X.coef + blockIdx.y*P_Simulation->Y.coef + blockIdx.z*P_Simulation->Z.coef + threadIdx.x*P_Simulation->W.coef;
	//if (index < P_Simulation->max)
		Tab_Iter[index] = iter;// index % 255;
	/*else
		printf("%d > %d", index, P_Simulation->max);*/
}

void printParameterer(struct_P_Simulation_T *P_Simulation)
{
	std::cout << "*****************  P_Simulation Config  *********************" << "\n";
	if (P_Simulation->W.step == 0.0f)
		std::cout << " W Fix = " << P_Simulation->W.start << "\n";
	else
		std::cout << " W start = " << P_Simulation->W.start << " to W end =" << P_Simulation->W.end << " by  W step =" << P_Simulation->W.step << "\n";

	if (P_Simulation->X.step == 0.0f)
		std::cout << " X Fix = " << P_Simulation->X.start << "\n";
	else
		std::cout << " X start = " << P_Simulation->X.start << " to X end =" << P_Simulation->X.end << " by  X step =" << P_Simulation->X.step << "\n";

	if (P_Simulation->Y.step == 0.0f)
		std::cout << " Y Fix = " << P_Simulation->Y.start << "\n";
	else
		std::cout << " Y start = " << P_Simulation->Y.start << " to Y end =" << P_Simulation->Y.end << " by  Y step =" << P_Simulation->Y.step << "\n";

	if (P_Simulation->Z.step == 0.0f)
		std::cout << " Z Fix = " << P_Simulation->Z.start << "\n";
	else
		std::cout << " Z start = " << P_Simulation->Z.start << " to Z end =" << P_Simulation->Z.end << " by  Z step =" << P_Simulation->Z.step << "\n";


	std::cout << "*****************  P_Simulation Config  *********************" << "\n";

}


struct_Stat_float_T compute(struct_P_All_T Config, bool state)
{
std::ofstream filetxt;
std::ofstream file;
	
int Tab_Histo[300];
int  Nbpoint_iter = 0;
int NoConfig = 0;
int NbConfig = max(1, Config.W.NbStep)* max(1, Config.X.NbStep)* max(1, Config.Y.NbStep)* max(1, Config.Z.NbStep);
//Stat
struct_Stat_float_T Stat;
for (int NoW = 0; NoW < max(1,Config.W.NbStep); NoW++)
{
	for (int NoX = 0; NoX < max(1, Config.X.NbStep); NoX++)
	{
		for (int NoY = 0; NoY < max(1, Config.Y.NbStep); NoY++)
		{
			for (int NoZ = 0; NoZ < max(1, Config.Z.NbStep); NoZ++)
			{
				NoConfig++;
				std::cout << "---------------------------------------------------\n";
				std::cout << "Config  " << NoConfig << " sur " << NbConfig << "\n";

				float W = NoW*Config.W.step + Config.W.start;
				float X = NoX*Config.X.step + Config.X.start;
				float Y = NoY*Config.Y.step + Config.Y.start;
				float Z = NoZ*Config.Z.step + Config.Z.start;

				int PasW = Config.NbPointByStep;
				if (Config.W.NbStep == 0)
					PasW = 1;

				int PasX = Config.NbPointByStep;
				if (Config.X.NbStep == 0)
					PasX = 1;

				int PasY = Config.NbPointByStep;
				if (Config.Y.NbStep == 0)
					PasY = 1;

				int PasZ = Config.NbPointByStep;
				if (Config.Z.NbStep == 0)
					PasZ = 1;

				//Taille de tableau
				int max =PasZ*PasY*PasX*PasW;

				if (Config.isShow)
					std::cout << "hipMallocManaged Config  -->  Start" << "\n";

				struct_P_Simulation_T *P_Simulation;
				int *Tab_Iter;
				// Allocate Unified Memory -- accessible from CPU or GPU
				hipMallocManaged(&P_Simulation, sizeof(struct_P_Simulation_T));
				hipMallocManaged(&Tab_Iter, max * sizeof(int));
				if (Config.isShow)
					std::cout << "hipMallocManaged Config  -->  End " << "\n";

				if (Config.isShow)
					std::cout << "P_Simulation Config  -->  Start" << "\n";
				// Pramatrage de W
				P_Simulation->W.start = W;
				P_Simulation->W.end = W + Config.W.step;
				P_Simulation->W.NbStep = PasW;
				if (PasW == 1)
					P_Simulation->W.step = 0.0f;
				else
					P_Simulation->W.step = (Config.W.step) / (PasW - 1);
				P_Simulation->W.coef = PasZ*PasY*PasX;

				// Pramatrage de X
				P_Simulation->X.start = X;
				P_Simulation->X.end = X + Config.X.step;
				P_Simulation->X.NbStep = PasX;
				if (PasX == 1)
					P_Simulation->X.step = 0.0f;
				else
					P_Simulation->X.step = (Config.X.step) / (PasX - 1);
				P_Simulation->X.coef = PasZ*PasY;

				// Pramatrage de Y
				P_Simulation->Y.start = Y;
				P_Simulation->Y.end = Y + Config.Y.step;
				P_Simulation->Y.NbStep = PasY;
				if (PasY == 1)
					P_Simulation->Y.step = 0.0f;
				else
					P_Simulation->Y.step = (Config.Y.step) / (PasY - 1);
				P_Simulation->Y.coef = PasZ;

				// Pramatrage de Z
				P_Simulation->Z.start = Z;
				P_Simulation->Z.end = Z + Config.Z.step;
				P_Simulation->Z.NbStep = PasZ;
				if (PasZ == 1)
					P_Simulation->Z.step = 0.0f;
				else
					P_Simulation->Z.step = (Config.Z.step) / (PasZ - 1);
				P_Simulation->Z.coef = 1;

				printParameterer(P_Simulation);

				//Stat Step

				Stat.Wstep = P_Simulation->W.step;
				Stat.Xstep = P_Simulation->X.step;
				Stat.Ystep = P_Simulation->Y.step;
				Stat.Zstep = P_Simulation->Z.step;


				//Parametrage Iter
				P_Simulation->Iter.max = Config.Iter.max;
				P_Simulation->Iter.min = Config.Iter.min;

				//Parametrage Power
				P_Simulation->power = Config.power;

				//Parametrage Rmax
				P_Simulation->rMax = Config.rMax;

				//Parametrage max
				P_Simulation->max = max;

				if (Config.isShow)
					std::cout << "P_Simulation Config  -->  End" << "\n";

				if (Config.isShow)
					std::cout << "Tab_Iter and Tab_Histo Init  -->  Start" << "\n";
				for (int i = 0; i < max; i++)
					Tab_Iter[i] = 0;

				for (int i = 0; i <= Config.Iter.max; i++)
					Tab_Histo[i] = 0;
				if (Config.isShow)
					std::cout << "Tab_Iter and Tab_Histo Init -->  End" << "\n";

				if (Config.isShow)
					std::cout << "Compude GPU -->  Start" << "\n";
				dim3 grid(PasX, PasY, PasZ);
				dim3 block(PasW, 1, 1);
				kernel << <grid, block >> >(P_Simulation, Tab_Iter);
				if (Config.isShow)
					std::cout << "Compude GPU -->  End" << "\n";


				if (Config.isShow)
					std::cout << "hipDeviceSynchronize-->  Start" << "\n";
				hipDeviceSynchronize();
				if (Config.isShow)
					std::cout << "hipDeviceSynchronize -->  End" << "\n";

				if (Config.isShow)
					std::cout << "Analyzer Simulation -->  Start" << "\n";
				Nbpoint_iter = 0;
				for (int i = 0; i < max; i++)
				{
					if (Tab_Iter[i] > 0)
						Nbpoint_iter++;
					Tab_Histo[Tab_Iter[i]]++;
				}
				if (Config.isShow)
				{
					std::cout << "Nb point Nbpoint_iter = " << Nbpoint_iter << "\n";
					std::cout << "Soit  :  " << (float)(Nbpoint_iter / ((float)max / 10000.0f)) / 100.0f << "%  soit " << Nbpoint_iter << "pt sur " << max << "pt \n";
					std::cout << "Analyzer Simulation -->  End" << "\n";
				}
				if (state == true)
				{
					if (Config.isShow)
						std::cout << "Write Histogram -->  Start" << "\n";
					file.open(Config.nameFile.histo, std::ofstream::out | std::ofstream::app);
					file << NoConfig << ";";
					for (int i = 0; i <= Config.Iter.max; i++)
						file << Tab_Histo[i] << ";";
					file << "\n";
					file.close();
					if (Config.isShow)
						std::cout << "Write Histogram -->  End" << "\n";
				}
				


				file.open(Config.nameFile.csv, std::ofstream::out | std::ofstream::app);
				filetxt.open(Config.nameFile.txt, std::ofstream::out | std::ofstream::app);


				for (int i = 0; i < max; i++)
				{
					int j = i;

					//W
					int iW = 0;
					if (PasW > 1)
					{
						iW = j / P_Simulation->W.coef;
					}
					//printf("index = %d  - Z Tempindex = %d \n", i, Tempindex);
					float w = (float)iW*P_Simulation->W.step + P_Simulation->W.start;
					// on retranche 
					j -= iW*P_Simulation->W.coef;

					//X
					int iX = 0;
					if (PasX > 1)
					{
						iX = j / P_Simulation->X.coef;
					}
					float x = (float)iX*P_Simulation->X.step + P_Simulation->X.start;
					// on retranche 
					j -= iX*P_Simulation->X.coef;

					//Y
					int iY = 0;
					if (PasY > 1)
					{
						iY = j / P_Simulation->Y.coef;
					}
					float y = (float)iY*P_Simulation->Y.step + P_Simulation->Y.start;
					// on retranche 
					j -= iY*P_Simulation->Y.coef;

					//Z
					int iZ = 0;
					if (PasZ > 1)
					{
						iZ = j / P_Simulation->Z.coef;
					}
					float z = (float)iZ*P_Simulation->Z.step + P_Simulation->Z.start;
					// on retranche 
					j -= iZ*P_Simulation->Z.coef;


					int iter = Tab_Iter[i];
					if ((iter >= Config.Iter.min && Config.Iter.isFix == 2) || (iter == Config.Iter.min && Config.Iter.isFix == 1))
					{
						int filter = Config.filter;
						if (state == true)
						{
							if (FilterQ_H(&filter, &iX, &iY, &iZ, &iW, iter, P_Simulation))
							{
								file << x << ";" << y << ";" << z << ";" << w << ";" << iter << ";\n";
								int NbDim = 0;
								if (PasW > 1)
								{
									filetxt << w << ";";
									NbDim++;
								}

								if (PasX > 1)
								{
									filetxt << x << ";";
									NbDim++;
								}

								if (PasY > 1)
								{
									filetxt << y << ";";
									NbDim++;
								}

								if (PasZ > 1)
								{
									filetxt << z << ";";
									NbDim++;
								}

								if (NbDim >= 3)
									filetxt << iter << "\n";
								else
									filetxt << ((float)iter) / ((float)Config.Iter.max)*3.0f << "\n";
							}
						}
						if (w > Stat.Wmax)
							Stat.Wmax = w;
						if (w < Stat.Wmin)
							Stat.Wmin = w;

						if (x > Stat.Xmax)
							Stat.Xmax = x;
						if (x < Stat.Xmin)
							Stat.Xmin = x;

						if (y > Stat.Ymax)
							Stat.Ymax = y;
						if (y < Stat.Ymin)
							Stat.Ymin = y;

						if (z > Stat.Zmax)
							Stat.Zmax = z;
						if (z < Stat.Zmin)
							Stat.Zmin = z;

						Stat.NbPoint++;
					}

				}
				file.close();
				filetxt.close();

				if (Config.isShow)
					std::cout << "Clear Mem + Reste  -->  Start" << "\n";
				hipFree(P_Simulation);
				hipFree(Tab_Iter);
				//hipDeviceReset();
				if (Config.isShow)
					std::cout << "Clear Mem + Reste  -->  End" << "\n";
			}
		}
	}
}

return Stat;

}


int main(int argc, char *argv[])
{
	struct_P_All_T Config;
    int state= menu( argc, argv, &Config);

	if (state != 1)
		return state;

	/********  Clear File ************/
	std::ofstream filetxt;
	filetxt.open(Config.nameFile.txt);
	filetxt.close();

	std::ofstream file;
	file.open(Config.nameFile.csv);
	file << "X;Y;Z;W;iter;\n";
	file.close();


	file.open(Config.nameFile.histo);
	file << "index;";
	for (int i = 0; i <= Config.Iter.max; i++)
		file << i << ";";
	file << "\n";
	file.close();

	//Affiche Stat
	FILE *fileman;
	char line[1000];
	fileman = fopen(Config.nameFile.stat, "r");
	if (fileman != NULL)
	{
		while (std::fgets(line, 1000, fileman))
		{
			std::cout << line;
		}
		fclose(fileman);
	}
	else
		std::cout << "Error  : "<< Config.nameFile.stat <<" no found" << "\n";
	hipSetDevice(Config.dev);


	struct_Stat_float_T Stat;

	if(Config.modeAM==false)
		Stat =compute(Config, true);
	else
	{
		bool Dim[] = {false,false ,false ,false };
		
		if (Config.W.isFix == 2)
			Dim[0] = true;
		if (Config.X.isFix == 2)
			Dim[1] = true;
		if (Config.Y.isFix == 2)
			Dim[2] = true;
		if (Config.Z.isFix == 2)
			Dim[3] = true;

		int NbDim = 0;

		for (int i = 0; i <= 3; i++)
		{
			if (Dim[i]==true)
			{
				NbDim++;
				switch (i)
				{
				case 0:
					Config.W.isFix = 2;
					Config.W.start = -20.0f;
					Config.W.end = 20.0f;
					Config.W.NbStep = 2;
					Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
					break;
				case 1:
					Config.X.isFix = 2;
					Config.X.start = -20.0f;
					Config.X.end = 20.0f;
					Config.X.NbStep = 2;
					Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
					break;
				case 2:
					Config.Y.isFix = 2;
					Config.Y.start = -20.0f;
					Config.Y.end = 20.0f;
					Config.Y.NbStep = 2;
					Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
					break;
				case 3:
					Config.Z.isFix = 2;
					Config.Z.start = -20.0f;
					Config.Z.end = 20.0f;
					Config.Z.NbStep = 2;
					Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
					break;
				default:
					break;
				}
			}	
		}
		//si et selement si il y a 3 dimentions
		if (NbDim == 3)
		{
			float pas = 10;
			float OldPas = 20;
			int nb_iter = 10;
			for (int iter = 0; iter <= nb_iter; iter++)
			{
				Stat = compute(Config,false);
				// rétro action
				for (int i = 0; i <= 3; i++)
				{
					if (Dim[i] == true)
					{
						switch (i)
						{
						case 0:
							if (Stat.Wmin == Config.W.start)
							{
								Config.W.start -= OldPas;
								if (nb_iter != i)
									Config.W.start += pas;
							}
							else if(nb_iter!=i)
								Config.W.start +=pas;

							if (Stat.Wmax == Config.W.end)
							{
								Config.W.end += OldPas;
								if (nb_iter != i)
									Config.W.end -= pas;
							}								
							else if(nb_iter != i)
								Config.W.end -= pas;
							Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
							break;
						case 1:
							if (Stat.Xmin == Config.X.start)
							{
								Config.X.start -= OldPas;
								if (nb_iter != i)
									Config.X.start += pas;
							}
							else if (nb_iter != i)
								Config.X.start += pas;
							if (Stat.Xmax == Config.X.end)
							{
								Config.X.end += OldPas;
								if (nb_iter != i)
									Config.X.end -= pas;
							}
							else if (nb_iter != i)
								Config.X.end -= pas;
							Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
							break;
						case 2:
							if (Stat.Ymin == Config.Y.start)
							{
								Config.Y.start -= OldPas;
								if (nb_iter != i)
									Config.Y.start += pas;
							}
							else if (nb_iter != i)
								Config.Y.start += pas;
							if (Stat.Ymax == Config.Y.end)
							{
								Config.Y.end += OldPas;
								if (nb_iter != i)
									Config.Y.end -= pas;
							}
							else if (nb_iter != i)
								Config.Y.end -= pas;
							Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
							break;
						case 3:
							if (Stat.Zmin == Config.Z.start)
							{
								Config.Z.start -= OldPas;
								if (nb_iter != i)
									Config.Z.start += pas;
							}
							else if (nb_iter != i)
								Config.Z.start += pas;
							if (Stat.Zmax == Config.Z.end)
							{
								Config.Z.end += OldPas;
								if (nb_iter != i)
									Config.Z.end -= pas;
							}
							else if (nb_iter != i)
								Config.Z.end -= pas;
							Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
							break;
						default:
							break;
						}
					}

				}

				file.open(Config.nameFile.stat, std::ofstream::out | std::ofstream::app);
				file << "Statistiques : \n";
				file << "				X min = " << Stat.Xmin << "\n";
				file << "				X max = " << Stat.Xmax << "\n";
				file << "				Y min = " << Stat.Ymin << "\n";
				file << "				Y max = " << Stat.Ymax << "\n";
				file << "				Z min = " << Stat.Zmin << "\n";
				file << "				Z max = " << Stat.Zmax << "\n";
				file << "				W min = " << Stat.Wmin << "\n";
				file << "				W max = " << Stat.Wmax << "\n";
				file << "				X step = " << Stat.Xstep << "\n";
				file << "				Y step = " << Stat.Ystep << "\n";
				file << "				Z step = " << Stat.Zstep << "\n";
				file << "				W step = " << Stat.Wstep << "\n";
				file << "				NbPoint plot = " << Stat.NbPoint << "\n";
				file.close();


				OldPas = pas;
				pas = pas / 2.0f;
			}

			for (int i = 0; i <= 3; i++)
			{
				if (Dim[i] == true)
				{
					switch (i)
					{
					case 0:
						Config.W.NbStep = 5;
						Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
						break;
					case 1:
						Config.X.NbStep = 5;
						Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
						break;
					case 2:
						Config.Y.NbStep = 5;
						Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
						break;
					case 3:
						Config.Z.NbStep = 5;
						Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
						break;
					default:
						break;
					}
				}

			}
			if(Stat.NbPoint!=0)
				Stat = compute(Config, true);
		}
		else
		{
			std::cout << "Error  mode: mode auto only 3 dim\n";
			return -1;
		}

	}
	
	file.open(Config.nameFile.stat, std::ofstream::out | std::ofstream::app);
	file << "Statistiques : \n";
	file << "				X min = " << Stat.Xmin << "\n";
	file << "				X max = " << Stat.Xmax << "\n";
	file << "				Y min = " << Stat.Ymin << "\n";
	file << "				Y max = " << Stat.Ymax << "\n";
	file << "				Z min = " << Stat.Zmin << "\n";
	file << "				Z max = " << Stat.Zmax << "\n";
	file << "				W min = " << Stat.Wmin << "\n";
	file << "				W max = " << Stat.Wmax << "\n";
	file << "				X step = " << Stat.Xstep << "\n";
	file << "				Y step = " << Stat.Ystep << "\n";
	file << "				Z step = " << Stat.Zstep << "\n";
	file << "				W step = " << Stat.Wstep << "\n";
	file << "				NbPoint plot = " << Stat.NbPoint << "\n";
	file.close();




	return 0;//Fin du programme
}