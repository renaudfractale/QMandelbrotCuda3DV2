#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //lib W10
#include ""//lib W10
#include <iostream> // prompt Output
#include <fstream> //File Output
#include <math.h> //lib mayh
#include <stdio.h> // lib stantard
//#include <hip/hip_fp16.h> // lib CUDA
#include <windows.h>
#include <fstream>
#include <string>
#define Dim_isFix 0;
#define Dim_end 20.0f;
#define Dim_start -20.0f;
#define Dim_NbStep 4;
#define Dim_step 10.0f;

#define ITER_MAX 255;
#define ITER_MIN 1;
#define ITER_isFix 0;

#define DEV 1;
#define FILTER 0;
#define POWER 2.0f;
#define ISSHOW 1;
#define RMAX 4.0f;

#define NBPOINTS 64;
//Strcture state

typedef struct 	struct_Stat_float {
	float Xmin = Dim_end
	float Xmax = Dim_start
	float Wmin = Dim_end
	float Wmax = Dim_start
	float Ymin = Dim_end
	float Ymax = Dim_start
	float Zmin = Dim_end
	float Zmax = Dim_start
	float Wstep = 0.0f;
	float Xstep = 0.0f;
	float Ystep = 0.0f;
	float Zstep = 0.0f;
	unsigned long NbPoint=0;
} struct_Stat_float_T;


// struct sur la gestion des dimensions
typedef struct 	struct_P_float {
	int isFix = Dim_isFix
	float start = Dim_start
	float end = Dim_end
	int NbStep = Dim_NbStep
	float step = Dim_step
	int coef = 1;
} struct_P_float_T;


typedef struct 	struct_Iter {
	int max = ITER_MAX
	int min = ITER_MIN
	int isFix = ITER_isFix
} struct_Iter_T;

typedef struct 	struct_FileName {
	char root[100];
	char txt[110];
	char csv[110];
	char stat[110];
	char histo[110];
	char stl[110];
} struct_FileName_T;

// struct sur la gestion param�tres d'entr�
typedef struct 	struct_P_All {
	struct_P_float_T X;
	struct_P_float_T Y;
	struct_P_float_T Z;
	struct_P_float_T W;
	struct_Iter_T Iter;
	int dev = DEV
	int filter=FILTER
	float power = POWER
	int isShow = ISSHOW
	struct_FileName_T nameFile;
	float rMax = RMAX
	int NbPointByStep = NBPOINTS
} struct_P_All_T;

typedef struct 	struct_P_Simulation {
	//Quaternions
	struct_P_float_T X;
	struct_P_float_T Y;
	struct_P_float_T Z;
	struct_P_float_T W;
	struct_Iter_T Iter;
	float rMax;
	//Parametrer variable systematique
	float power;
	int max;
} struct_P_Simulation_T;

typedef struct 	struct_Q {
	float x;
	float y;
	float z;
	float w;
} struct_Q_T;

//__managed__  struct_P_Simulation_T *P_Simulation;
//__managed__  int *Tab_Iter;

__host__  void CreateQ_By_float_H(struct_Q_T *out, float x, float y, float z, float w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

__host__  float  Get_QNorm_H(struct_Q_T *Q)
{
	return sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
}

__host__ void Get_QPow_H(struct_Q_T *Q, float pow)
{
	float A = Get_QNorm_H(Q);
	float theta = 0.0f;
	float B = 0.0f;
	float R = 0.0f;
	if (pow > 0.0f && A>0.000001f)
	{
		float coef = 1.0f;
		if (A<1.0f)
		{
			//printf("%f *******\n", A);
			coef = 1 / A;
			Q->x *= coef;
			Q->y *= coef;
			Q->z *= coef;
			Q->z *= coef;

		}
		A = Get_QNorm_H(Q);
		//printf("%f +++++++++\n", A);
		theta = acosf(Q->w / A)*pow;
		B = sqrt(A*A - Q->w*Q->w);
		R = exp2f(logf(A / coef)* pow);
		Q->x = R*sinf(theta)*(Q->x / B);
		Q->y = R*sinf(theta)*(Q->y / B);
		Q->z = R*sinf(theta)*(Q->z / B);
		Q->z = R*cosf(theta);

	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0f;
		Q->x = 0.0f;
		Q->y = 0.0f;
		Q->z = 0.0f;

	}
}

__host__ int  GetQIter_H(struct_P_Simulation_T *P_Simulation_DEVICE, int  *x_filter, int  *y_filter, int *z_filter, int *w_filter)
{
	//int Tempindex = 0;
	struct_Q_T Q_Current;
	float w, x, y, z;
	int iter_computed;
	//X
	x = ((float)*x_filter)*P_Simulation_DEVICE->X.step + P_Simulation_DEVICE->X.start;
	//Y
	y = ((float)*y_filter)*P_Simulation_DEVICE->Y.step + P_Simulation_DEVICE->Y.start;
	//Z
	z = ((float)*z_filter)*P_Simulation_DEVICE->Z.step + P_Simulation_DEVICE->Z.start;
	//W
	w = ((float)*w_filter)*P_Simulation_DEVICE->W.step + P_Simulation_DEVICE->W.start;



	CreateQ_By_float_H(&Q_Current, x, y, z, w);

	for (iter_computed = 0; iter_computed <= P_Simulation_DEVICE->Iter.max; iter_computed++)
	{
		Get_QPow_H(&Q_Current, P_Simulation_DEVICE->power);
		Q_Current.x += x;
		Q_Current.y += y;
		Q_Current.z += z;
		Q_Current.w += w;

		if (Get_QNorm_H(&Q_Current) > P_Simulation_DEVICE->rMax)
		{
			if (iter_computed > 0)
				iter_computed--;
			return iter_computed;
		}
	}
	if (iter_computed > 0)
		iter_computed--;
	return iter_computed;
}
__host__ bool  FilterQ_H(int *Filter, int *Nx, int *Ny, int *Nz, int *Nw, int iter,
	struct_P_Simulation_T *P_Simulation)
{
	if (*Filter == 0)
		return true;
	int iter_computed = 0;

	int pasx = 1;
	if (P_Simulation->X.NbStep == 1)
		pasx = 0;

	int pasy = 1;
	if (P_Simulation->Y.NbStep == 1)
		pasy = 0;

	int pasz = 1;
	if (P_Simulation->Z.NbStep == 1)
		pasz = 0;

	int pasw = 1;
	if (P_Simulation->W.NbStep == 1)
		pasw = 0;


	for (int x_filter = *Nx - pasx ; x_filter <= *Nx + pasx; x_filter++)
	{
		for (int y_filter = *Ny - pasy; y_filter <= *Ny + pasy; y_filter++)
		{
			for (int z_filter = *Nz - pasz; z_filter <= *Nz + pasz; z_filter++)
			{
				for (int w_filter = *Nw - pasw; w_filter <= *Nw + pasw; w_filter++)
				{
					iter_computed = GetQIter_H(P_Simulation, &x_filter, &y_filter, &z_filter, &w_filter);
					if (*Filter == 1)
					{
						if (iter_computed != iter)
							return true;
					}
					else //filter==2
					{
						if (iter_computed == 0)
							return true;
					}
				}
			}
		}
	}
	return false;
}


__device__  void CreateQ_By_float(struct_Q_T *out, float x, float y, float z, float w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

__device__  float  Get_QNorm(struct_Q_T *Q)
{
	return sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
}

__device__ void Get_QPow(struct_Q_T *Q, float pow)
{
	float A = Get_QNorm(Q);
	float theta = 0.0f;
	float B = 0.0f;
	float R = 0.0f;
	if (pow > 0.0f && A>0.000001f)
	{
		float coef = 1.0f;
		if (A<1.0f)
		{
			//printf("%f *******\n", A);
			coef = 1 / A;
			Q->x *= coef;
			Q->y *= coef;
			Q->z *= coef;
			Q->z *= coef;

		}
		A = Get_QNorm(Q);
		//printf("%f +++++++++\n", A);
		theta = acosf(Q->w / A)*pow;
		B = sqrt(A*A - Q->w*Q->w);
		R = exp2f(logf(A / coef)* pow);
		Q->x = R*sinf(theta)*(Q->x / B);
		Q->y = R*sinf(theta)*(Q->y / B);
		Q->z = R*sinf(theta)*(Q->z / B);
		Q->z = R*cosf(theta);

	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0f;
		Q->x = 0.0f;
		Q->y = 0.0f;
		Q->z = 0.0f;

	}
}
// CUDA kernel to Compute itermax of quaternion
__global__ void kernel(const struct_P_Simulation_T *P_Simulation, int *Tab_Iter)
{
	//int Tempindex = 0;
	struct_Q_T Q_Current;
	float w, x, y, z;
	int iter = 0;
	//X
	x = ((float)blockIdx.x)*P_Simulation->X.step + P_Simulation->X.start;
	//Y
	y = ((float)blockIdx.y)*P_Simulation->Y.step + P_Simulation->Y.start;
	//Z
	z = ((float)blockIdx.z)*P_Simulation->Z.step + P_Simulation->Z.start;
	//W
	w = ((float)threadIdx.x)*P_Simulation->W.step + P_Simulation->W.start;

	CreateQ_By_float(&Q_Current, x, y, z, w);

	for (iter = 0; iter <= P_Simulation->Iter.max; iter++)
	{
		Get_QPow(&Q_Current, P_Simulation->power);
		Q_Current.x += x;
		Q_Current.y += y;
		Q_Current.z += z;
		Q_Current.w += w;

		if (Get_QNorm(&Q_Current) > P_Simulation->rMax)
			goto Fin;
	}
Fin:
	if (iter > 0)
		iter--;
	int index = blockIdx.x*P_Simulation->X.coef + blockIdx.y*P_Simulation->Y.coef + blockIdx.z*P_Simulation->Z.coef + threadIdx.x*P_Simulation->W.coef;
	if (index < P_Simulation->max)
		Tab_Iter[index] = iter;// index % 255;
	else
		printf("%d > %d", index, P_Simulation->max);
}
int main(int argc, char *argv[])
{
	//Config
		struct_P_All_T Config;
		strcpy(Config.nameFile.root, "O");
		if (argc == 1)
		{
			Config.W.end = -0.3375f;
			Config.W.start = -0.3375f;
			Config.W.step = 1.0f;
			Config.W.isFix = 1;
			Config.W.NbStep = 1;


			Config.X.isFix = 2;
			Config.X.start = -10.0f;
			Config.X.end = 10.0f;

			Config.Y.isFix = 2;
			Config.Y.start = -10.0f;
			Config.Y.end = 10.0f;

			Config.Z.isFix = 2;
			Config.Z.start = -10.0f;
			Config.Z.end = 10.0f;

			Config.filter = 2;

			//Config.Iter.isFix = 1;
			//Config.Iter.max = 2;
			//Config.Iter.min = 2;
			//Config.rMax = 2;
		}


	//Stat
		struct_Stat_float_T Stat;

	//Arg Help
		char Str_H[] = "-h";
		char Str_Help[] = "--help";
	
	//Arg X
		char Str_xFix[] = "-x";
		char Str_xMax[] = "-xmax";
		char Str_xMin[] = "-xmin";
		char Str_xNbStep[] = "-xNbStep";
	//Arg W
		char Str_wFix[] = "-w";
		char Str_wMax[] = "-wmax";
		char Str_wMin[] = "-wmin";
		char Str_wNbStep[] = "-wNbStep";
	//Arg Y
		char Str_yFix[] = "-y";
		char Str_yMax[] = "-ymax";
		char Str_yMin[] = "-ymin";
		char Str_yNbStep[] = "-yNbStep";
	//Arg Z
		char Str_zFix[] = "-z";
		char Str_zMax[] = "-zmax";
		char Str_zMin[] = "-zmin";
		char Str_zNbStep[] = "-zNbStep";
	//Arg Iter
		char Str_IterFix[] = "-iter";
		char Str_IterMax[] = "-iterMax";
		char Str_IterMin[] = "-iterMin";
	//Arg Dev
		char Str_dev[] = "-device";
	//Arg Filter
		char Str_filter[] = "-filter";
	//Arg Power
		char Str_power[] = "-power";
	//Arg IsShow
		char Str_IsShow[] = "-isShow";
	//Arg Output File
		char Str_Out[] = "-o";
	//Arg Rmax
		char Str_rMax[] = "-rMax";
	//Si il y a des Arguments
	if (argc > 1)
	{
		// d�dection la commande -help pu -h
		for (int i = 1; i < argc; i++)
		{
			if (strcmp(argv[i], Str_Help) == 0 || strcmp(argv[i], Str_H) == 0)
			{
				//Affiche Help
				FILE *fileman;
				char line[1000];
				fileman = fopen("man", "r");
				if (fileman != NULL)
				{
					while (std::fgets(line, 1000, fileman))
					{
						std::cout << line;
					}
					fclose(fileman);
					return 0; //Fin du programme
				}
				else
				{
					std::cout << "Error  : man no found" << "\n";
					return -1;
				}			
			}
		}
		//Verification : si nb d'arguments est paire --> erreur
		if (argc % 2 == 0)
		{
			std::cout << "Error 00 : Argument impaire" << "\n";
			return -1;
		}
		else //: si nb d'arguments est impaire --> fonctionement normale
		{
			for (int i = 1; i < argc; i += 2)
			{
				std::cout << "Analyse du couple d'arguments :  " << argv[i] << " " << argv[i + 1] << "\n";
				if (strcmp(argv[i], Str_xFix) == 0)
				{
					if (Config.X.isFix == 0) //Si premier config
					{
						Config.X.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xFix << ": value is not float " << "\n";
							return -1;
						}
						Config.X.start = value;
						Config.X.end = value;
						Config.X.NbStep = 1;
						Config.X.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_xFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wFix) == 0)
				{
					if (Config.W.isFix == 0) //Si premier config
					{
						Config.W.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wFix << ": value is not float " << "\n";
							return -1;
						}
						Config.W.start = value;
						Config.W.end = value;
						Config.W.NbStep = 1;
						Config.W.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_wFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yFix) == 0)
				{
					if (Config.Y.isFix == 0) //Si premier config
					{
						Config.Y.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yFix << ": value is not float " << "\n";
							return -1;
						}
						Config.Y.start = value;
						Config.Y.end = value;
						Config.Y.NbStep = 1;
						Config.Y.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_yFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zFix) == 0)
				{
					if (Config.Z.isFix == 0) //Si premier config
					{
						Config.Z.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zFix << ": value is not float " << "\n";
							return -1;
						}
						Config.Z.start = value;
						Config.Z.end = value;
						Config.Z.NbStep = 1;
						Config.Z.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_zFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xMax) == 0)
				{
					if (Config.X.isFix == 0 || Config.X.isFix == 2)
					{
						Config.X.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xMax << ": value is not float " << "\n";
							return -1;
						}
						Config.X.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wMax) == 0)
				{
					if (Config.W.isFix == 0 || Config.W.isFix == 2)
					{
						Config.W.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wMax << ": value is not float " << "\n";
							return -1;
						}
						Config.W.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yMax) == 0)
				{
					if (Config.Y.isFix == 0 || Config.Y.isFix == 2)
					{
						Config.Y.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yMax << ": value is not float " << "\n";
							return -1;
						}
						Config.Y.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zMax) == 0)
				{
					if (Config.Z.isFix == 0 || Config.Z.isFix == 2)
					{
						Config.Z.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zMax << ": value is not float " << "\n";
							return -1;
						}
						Config.Z.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_zMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xMin) == 0)
				{
					if (Config.X.isFix == 0 || Config.X.isFix == 2)
					{
						Config.X.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xMin << ": value is not float " << "\n";
							return -1;
						}
						Config.X.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wMin) == 0)
				{
					if (Config.W.isFix == 0 || Config.W.isFix == 2)
					{
						Config.W.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wMin << ": value is not float " << "\n";
							return -1;
						}
						Config.W.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yMin) == 0)
				{
					if (Config.Y.isFix == 0 || Config.Y.isFix == 2)
					{
						Config.Y.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yMin << ": value is not float " << "\n";
							return -1;
						}
						Config.Y.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zMin) == 0)
				{
					if (Config.Z.isFix == 0 || Config.Z.isFix == 2)
					{
						Config.Z.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zMin << ": value is not float " << "\n";
							return -1;
						}
						Config.Z.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_zMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xNbStep) == 0)
				{
					if (Config.X.isFix == 0 || Config.X.isFix == 2)
					{
						Config.X.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_xNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.X.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wNbStep) == 0)
				{
					if (Config.W.isFix == 0 || Config.W.isFix == 2)
					{
						Config.W.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_wNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.W.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yNbStep) == 0)
				{
					if (Config.Y.isFix == 0 || Config.Y.isFix == 2)
					{
						Config.Y.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_yNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.Y.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zNbStep) == 0)
				{
					if (Config.Z.isFix == 0 || Config.Z.isFix == 2)
					{
						Config.Z.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_yNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.Z.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterFix) == 0)
				{
					if (Config.Iter.isFix == 0) //Si premier config
					{
						Config.Iter.isFix = 1;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_IterFix << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config.Iter.max = value;
						Config.Iter.min = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterMax) == 0)
				{
					if (Config.Iter.isFix == 0 || Config.Iter.isFix == 2) //Si premier config
					{
						Config.Iter.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_IterMax << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config.Iter.max = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterMin) == 0)
				{
					if (Config.Iter.isFix == 0 || Config.Iter.isFix == 2) //Si premier config
					{
						Config.Iter.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_IterMin << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config.Iter.min = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_dev) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_dev << ": value is not int" << "\n";
						return -1;
					}
					Config.dev = value;
				}
				else if (strcmp(argv[i], Str_filter) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_filter << ": value is not int" << "\n";
						return -1;
					}
					Config.filter = value;
				}
				else if (strcmp(argv[i], Str_power) == 0)
				{
					float value = (float)atof(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_filter << ": value is not float" << "\n";
						return -1;
					}
					Config.power = value;
				}
				else if (strcmp(argv[i], Str_IsShow) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_IsShow << ": value is not int" << "\n";
						return -1;
					}
					Config.isShow = value;
				}
				else if (strcmp(argv[i], Str_rMax) == 0)
				{
					float value = (float)atof(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_rMax << ": value is not float" << "\n";
						return -1;
					}
					Config.rMax = value;
				}
				else if (strcmp(argv[i], Str_Out) == 0)
				{
					if (strlen(argv[i + 1])<100)
						strcpy(Config.nameFile.root, argv[i + 1]);
					else
					{
						std::cout << "Error 04 strlen fileOutput must be inf to 100 signe \n";
						return -1;
					}
				}
				else
				{
					std::cout << "Warning 05  Arg not know : " << argv[i] << " " << argv[i + 1] << "\n";
				}
			}
		}
	}
	bool IsErrors = false;
	// Verification W
	if (Config.W.isFix == 1 && Config.W.start == Config.W.end)
	{
		;//OK
	}
	else if(Config.W.isFix == 2 || Config.W.isFix == 0)
	{
		Config.W.isFix = 2;
		if (Config.W.start < Config.W.end && Config.W.NbStep>1)
		{
			Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
		}
		else
		{
			std::cout << "Error W  :  wmax must be sup wmin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error W  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification X
	if (Config.X.isFix == 1 && Config.X.start == Config.X.end)
	{
		;//OK
	}
	else if (Config.X.isFix == 2 || Config.X.isFix == 0)
	{
		Config.X.isFix = 2; 
		if (Config.X.start < Config.X.end && Config.X.NbStep>1)
		{
			Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
		}
		else
		{
			std::cout << "Error X  :  xmax must be sup xmin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error X  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification Y
	if (Config.Y.isFix == 1 && Config.Y.start == Config.Y.end)
	{
		;//OK
	}
	else if (Config.Y.isFix == 2 || Config.Y.isFix == 0)
	{
		Config.Y.isFix = 2; 
		if (Config.Y.start < Config.Y.end && Config.Y.NbStep>1)
		{
			Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
		}
		else
		{
			std::cout << "Error Y  :  ymax must be sup ymin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error Y  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification Z
	if (Config.Z.isFix == 1 && Config.Z.start == Config.Z.end)
	{
		;//OK
	}
	else if (Config.Z.isFix == 2 || Config.Z.isFix == 0)
	{
		Config.Z.isFix = 2; 
		if (Config.Z.start < Config.Z.end && Config.Z.NbStep>1)
		{
			Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
		}
		else
		{
			std::cout << "Error Z  :  zmax must be sup zmin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error Z  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification Iter
	if (Config.Iter.isFix == 1)
	{
		if (Config.Iter.min == Config.Iter.max && Config.Iter.max > 0)
		{
			; //OK
		}
		else
		{
			std::cout << "Error Iter  :  value must be sup at 0\n";
			IsErrors=true;
		}
	}
	else if (Config.Iter.isFix == 2 || Config.Iter.isFix == 0)
	{
		Config.Iter.isFix = 2;
		if (Config.Iter.min < Config.Iter.max)
		{
			; // OK
		}
		else
		{
			std::cout << "Error Iter  :  max must be sup min\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error Iter  :  Error unknow\n";
		IsErrors=true;
	}
	// V�rification dev
	int count;
	hipGetDeviceCount(&count);
	if (Config.dev >= 0 && Config.dev < count)
	{
		; //OK
	}
	else
	{
		std::cout << "Error dev :  dev must be between 0 and "<< count -1 <<"\n";
		IsErrors=true;
	}
	//Verification filter
	if (Config.filter >= 0 && Config.filter <= 2)
	{
		if (Config.Iter.isFix == 1 && Config.filter == 2)
			Config.filter = 1;
	}
	else
	{
		std::cout << "Error filter :  filter must be between 0 and 2\n";
		IsErrors=true;
	}
	//Verification power
	if (Config.power >= 2.0f && Config.power <= 50.0f)
	{
		; //OK
	}
	else
	{
		std::cout << "Error power :  power must be between 2.0 and 50.0\n";
		IsErrors=true;
	}
	//Verification IsShow
	if (Config.isShow == 0 || Config.isShow == 1)
	{
		; //OK
	}
	else
	{
		std::cout << "Error isShow :  isShow must be between 0 and 1 \n";
		IsErrors=true;
	}
	//Verification rMax
	if (Config.rMax >0.0f)
	{
		; //OK
	}
	else
	{
		std::cout << "Error rMax :  rMax must be sup 0.0 \n";
		IsErrors = true;
	}
	if (IsErrors)
		return -1;

	// creation des fichiers :
	strcpy(Config.nameFile.csv, Config.nameFile.root);
	strcpy(Config.nameFile.histo, Config.nameFile.root);
	strcpy(Config.nameFile.stat, Config.nameFile.root);
	strcpy(Config.nameFile.stl, Config.nameFile.root);
	strcpy(Config.nameFile.txt, Config.nameFile.root);
	
	strcat(Config.nameFile.csv, ".csv");
	strcat(Config.nameFile.histo, ".histo");
	strcat(Config.nameFile.stat, ".stat");
	strcat(Config.nameFile.stl, ".stl");
	strcat(Config.nameFile.txt, ".txt");

	//Affichage de la config
	std::ofstream file;
	file.open(Config.nameFile.stat);
	file << "Parameters Current : " << "\n";
	file << "				W_start = " << Config.W.start << ", W_end = " << Config.W.end << ", W_Step = " << Config.W.step << ", W_NbStep = " << Config.W.NbStep << "\n";
	file << "				X_start = " << Config.X.start << ", X_end = " << Config.X.end << ", X_Step = " << Config.X.step << ", X_NbStep = " << Config.X.NbStep << "\n";
	file << "				Y_start = " << Config.Y.start << ", Y_end = " << Config.Y.end << ", Y_Step = " << Config.Y.step << ", Y_NbStep = " << Config.Y.NbStep << "\n";
	file << "				Z_start = " << Config.Z.start << ", Z_end = " << Config.Z.end << ", Z_Step = " << Config.Z.step << ", Z_NbStep = " << Config.Z.NbStep << "\n";
	file << "				Root FileOutput = " << Config.nameFile.root << "\n";
	file << "				iterMax = " << Config.Iter.max << "\n";
	file << "				iterMin = " << Config.Iter.min << "\n";
	file << "				rMax = " << Config.rMax << "\n";
	file << "				Filter = " << Config.filter<< "\n";
	file << "				Power = " << Config.power << "\n";
	file << "				dev = " << Config.dev << "\n";
	file << "				IsShow = " << Config.isShow << "\n";
	file << "				NbPoints per step = " << Config.NbPointByStep << "\n";
	file << "				ouput File :  " << Config.nameFile.root << "\n";
	file << "cmd for use this configuration: " << "\n";
	file << "               " << argv[0] << "  ";
	if (Config.W.isFix == 1)
		file << Str_wFix << " " << Config.W.start << " ";
	else
	{
		file << Str_wMin << " " << Config.W.start << " ";
		file << Str_wMax << " " << Config.W.end << " ";
		file << Str_wNbStep << " " << Config.W.NbStep << " ";
	}
	if (Config.X.isFix == 1)
		file << Str_xFix << " " << Config.X.start << " ";
	else
	{
		file << Str_xMin << " " << Config.X.start << " ";
		file << Str_xMax << " " << Config.X.end << " ";
		file << Str_xNbStep << " " << Config.X.NbStep << " ";
	}
	if (Config.Y.isFix == 1)
		file << Str_yFix << " " << Config.Y.start << " ";
	else
	{
		file << Str_yMin << " " << Config.Y.start << " ";
		file << Str_yMax << " " << Config.Y.end << " ";
		file << Str_yNbStep << " " << Config.Y.NbStep << " ";
	}
	if (Config.Z.isFix == 1)
		file << Str_zFix << " " << Config.Z.start << " ";
	else
	{
		file << Str_zMin << " " << Config.Z.start << " ";
		file << Str_zMax << " " << Config.Z.end << " ";
		file << Str_zNbStep << " " << Config.Z.NbStep << " ";
	}
	if (Config.Iter.isFix == 1)
		file << Str_IterFix << " " << Config.Iter.max << " ";
	else
	{
		file << Str_IterMin << " " << Config.Iter.min << " ";
		file << Str_IterMax << " " << Config.Iter.max << " ";
	}
	file << Str_dev << " " << Config.dev << " ";
	file << Str_filter << " " << Config.filter << " ";
	file << Str_power << " " << Config.power << " ";
	file << Str_IsShow << " " << Config.isShow << " ";
	file << Str_rMax << " " << Config.rMax << " ";
	file << Str_Out << " " << Config.nameFile.root << " ";

	file << "\n";
	file.close();
	/********  Clear File ************/
	std::ofstream filetxt;
	filetxt.open(Config.nameFile.txt);
	filetxt.close();

	file.open(Config.nameFile.csv);
	file << "X;Y;Z;W;iter;\n";
	file.close();


	file.open(Config.nameFile.histo);
	file << "index;";
	for (int i = 0; i <= Config.Iter.max; i++)
		file << i << ";";
	file << "\n";
	file.close();

	//Affiche Help
	FILE *fileman;
	char line[1000];
	fileman = fopen(Config.nameFile.stat, "r");
	if (fileman != NULL)
	{
		while (std::fgets(line, 1000, fileman))
		{
			std::cout << line;
		}
		fclose(fileman);
	}
	else
		std::cout << "Error  : "<< Config.nameFile.stat <<" no found" << "\n";

	int Tab_Histo[300];
	int  Nbpoint_iter = 0;
	hipSetDevice(Config.dev);
	int NoConfig = 0;
	int NbConfig = Config.W.NbStep*Config.X.NbStep*Config.Y.NbStep*Config.Z.NbStep;

	for (int NoW = 0; NoW < Config.W.NbStep; NoW++)
	{
		for (int NoX = 0; NoX < Config.X.NbStep; NoX++)
		{
			for (int NoY = 0; NoY < Config.Y.NbStep; NoY++)
			{
				for (int NoZ = 0; NoZ < Config.Z.NbStep; NoZ++)
				{
					NoConfig++;
					std::cout << "---------------------------------------------------\n";
					std::cout << "Config  " << NoConfig << " sur " << NbConfig << "\n";

					float W = NoW*Config.W.step + Config.W.start;
					float X = NoX*Config.X.step + Config.X.start;
					float Y = NoY*Config.Y.step + Config.Y.start;
					float Z = NoZ*Config.Z.step + Config.Z.start;

					int PasW = Config.NbPointByStep;
					if (Config.W.NbStep == 1)
						PasW = 1;

					int PasX = Config.NbPointByStep;
					if (Config.X.NbStep == 1)
						PasX = 1;

					int PasY = Config.NbPointByStep;
					if (Config.Y.NbStep == 1)
						PasY = 1;

					int PasZ = Config.NbPointByStep;
					if (Config.Z.NbStep == 1)
						PasZ = 1;

					//Taille de tableau
					int max = PasZ*PasY*PasX*PasW;

					if (Config.isShow)
						std::cout << "hipMallocManaged Config  -->  Start" << "\n";

					struct_P_Simulation_T *P_Simulation;
					int *Tab_Iter;
					// Allocate Unified Memory -- accessible from CPU or GPU
					hipMallocManaged(&P_Simulation, sizeof(struct_P_Simulation_T));
					hipMallocManaged(&Tab_Iter, max * sizeof(int));
					if (Config.isShow)
						std::cout << "hipMallocManaged Config  -->  End " << "\n";

					if (Config.isShow)
						std::cout << "P_Simulation Config  -->  Start" << "\n";
					// Pramatrage de W
					P_Simulation->W.start = W;
					P_Simulation->W.end = W+ Config.W.step;
					P_Simulation->W.NbStep = PasW;
					if(PasW ==1)
						P_Simulation->W.step = 0.0f;
					else
						P_Simulation->W.step = (Config.W.step) / (PasW-1);
					P_Simulation->W.coef = PasX*PasY*PasZ;

					// Pramatrage de X
					P_Simulation->X.start = X;
					P_Simulation->X.end = X + Config.X.step;
					P_Simulation->X.NbStep = PasX;
					if (PasX == 1)
						P_Simulation->X.step = 0.0f;
					else
						P_Simulation->X.step = (Config.X.step) / (PasX-1);
					P_Simulation->X.coef = PasY*PasZ;

					// Pramatrage de Y
					P_Simulation->Y.start = Y;
					P_Simulation->Y.end = Y + Config.Y.step;
					P_Simulation->Y.NbStep = PasY;
					if (PasY == 1)
						P_Simulation->Y.step = 0.0f;
					else
						P_Simulation->Y.step = (Config.Y.step) / (PasY - 1);
					P_Simulation->Y.coef = PasZ;

					// Pramatrage de Z
					P_Simulation->Z.start = Z;
					P_Simulation->Z.end = Z + Config.Z.step;
					P_Simulation->Z.NbStep = PasZ;
					if (PasZ == 1)
						P_Simulation->Z.step = 0.0f;
					else
						P_Simulation->Z.step = (Config.Z.step) / (PasZ - 1);
					P_Simulation->Z.coef = 1;

					//Stat Step

					Stat.Wstep = P_Simulation->W.step;
					Stat.Xstep = P_Simulation->X.step;
					Stat.Ystep = P_Simulation->Y.step;
					Stat.Zstep = P_Simulation->Z.step;


					//Parametrage Iter
					P_Simulation->Iter.max = Config.Iter.max;
					P_Simulation->Iter.min = Config.Iter.min;

					//Parametrage Power
					P_Simulation->power = Config.power;

					//Parametrage Rmax
					P_Simulation->rMax = Config.rMax;

					//Parametrage max
					P_Simulation->max = max;

					if (Config.isShow)
						std::cout << "P_Simulation Config  -->  End" << "\n";

					if (Config.isShow)
						std::cout << "Tab_Iter and Tab_Histo Init  -->  Start" << "\n";
					for (int i = 0; i < max; i++)
						Tab_Iter[i] = 0;

					for (int i = 0; i <= Config.Iter.max; i++)
						Tab_Histo[i] = 0;
					if (Config.isShow)
						std::cout << "Tab_Iter and Tab_Histo Init -->  End" << "\n";

					if (Config.isShow)
						std::cout << "Compude GPU -->  Start" << "\n";
					dim3 grid(PasX, PasY, PasZ);
					dim3 block(PasW, 1, 1);
					kernel << <grid, block >> >(P_Simulation, Tab_Iter);
					if (Config.isShow)
						std::cout << "Compude GPU -->  End" << "\n";


					if (Config.isShow)
						std::cout << "hipDeviceSynchronize-->  Start" << "\n";
					hipDeviceSynchronize();
					if (Config.isShow)
						std::cout << "hipDeviceSynchronize -->  End" << "\n";

					if (Config.isShow)
						std::cout << "Analyzer Simulation -->  Start" << "\n";
					Nbpoint_iter = 0;
					for (int i = 0; i < max; i++)
					{
						if (Tab_Iter[i] > 0)
							Nbpoint_iter++;
						Tab_Histo[Tab_Iter[i]]++;
					}
					if (Config.isShow)
					{
						std::cout << "Nb point Nbpoint_iter = " << Nbpoint_iter << "\n";
						std::cout << "Soit  :  " << (float)(Nbpoint_iter / ((float)max / 10000.0f)) / 100.0f << "%  soit " << Nbpoint_iter << "pt sur " << max << "pt \n";
						std::cout << "Analyzer Simulation -->  End" << "\n";
					}

					if (Config.isShow)
						std::cout << "Write Histogram -->  Start" << "\n";
					file.open(Config.nameFile.histo, std::ofstream::out | std::ofstream::app);
					file << NoConfig << ";";
					for (int i = 0; i <= Config.Iter.max; i++)
						file << Tab_Histo[i] << ";";
					file << "\n";
					file.close();
					if (Config.isShow)
						std::cout << "Write Histogram -->  End" << "\n";
					
					
					file.open(Config.nameFile.csv, std::ofstream::out | std::ofstream::app);
					filetxt.open(Config.nameFile.txt , std::ofstream::out | std::ofstream::app);

					
					for (int i = 0; i < max; i++)
					{
						int j = i;

						//W
						int iW = 0;
						if (PasW > 1)
						{
							iW = j / P_Simulation->W.coef;
						}
						//printf("index = %d  - Z Tempindex = %d \n", i, Tempindex);
						float w = (float)iW*P_Simulation->W.step + P_Simulation->W.start;
						// on retranche 
						j -= iW*P_Simulation->W.coef;

						//X
						int iX = 0;
						if (PasX > 1)
						{
							iX = j / P_Simulation->X.coef;
						}
						float x = (float)iX*P_Simulation->X.step + P_Simulation->X.start;
						// on retranche 
						j -= iX*P_Simulation->X.coef;

						//Y
						int iY = 0;
						if (PasY > 1)
						{
							iY = j / P_Simulation->Y.coef;
						}
						float y = (float)iY*P_Simulation->Y.step + P_Simulation->Y.start;
						// on retranche 
						j -= iY*P_Simulation->Y.coef;

						//Z
						int iZ = 0;
						if (PasZ > 1)
						{
							iZ = j / P_Simulation->Z.coef;
						}
						float z = (float)iZ*P_Simulation->Z.step + P_Simulation->Z.start;
						// on retranche 
						j -= iZ*P_Simulation->Z.coef;


						int iter = Tab_Iter[i];
						if ((iter >= Config.Iter.min && Config.Iter.isFix==2) || (iter == Config.Iter.min && Config.Iter.isFix == 1))
						{
							int filter = Config.filter;
							if (FilterQ_H(&filter, &iX, &iY, &iZ, &iW, iter, P_Simulation))
							{
								file << x << ";" << y << ";" << z << ";" << w << ";" << iter << ";\n";
								int NbDim = 0;
								if (PasW > 1)
								{
									filetxt << w << ";";
									NbDim++;
								}
									
								if (PasX > 1)
								{
									filetxt << x << ";";
									NbDim++;
								}
									
								if (PasY > 1)
								{
									filetxt << y << ";"; 
									NbDim++;
								}
									
								if (PasZ > 1)
								{
									filetxt << z << ";"; 
									NbDim++;
								}
								
								if(NbDim>=3)
									filetxt << iter << "\n";
								else
									filetxt << ((float)iter)/((float)Config.Iter.max)*3.0f << "\n";
							}
							if (w > Stat.Wmax)
								Stat.Wmax = w;
							if (w < Stat.Wmin)
								Stat.Wmin = w;

							if (x > Stat.Xmax)
								Stat.Xmax = x;
							if (x < Stat.Xmin)
								Stat.Xmin = x;

							if (y > Stat.Ymax)
								Stat.Ymax = y;
							if (y < Stat.Ymin)
								Stat.Ymin = y;

							if (z > Stat.Zmax)
								Stat.Zmax = z;
							if (z < Stat.Zmin)
								Stat.Zmin = z;

							Stat.NbPoint++;
							}

					}
					file.close();
					filetxt.close();

					if (Config.isShow)
						std::cout << "Clear Mem + Reste  -->  Start" << "\n";
					hipFree(P_Simulation);
					hipFree(Tab_Iter);
					//hipDeviceReset();
					if (Config.isShow)
						std::cout << "Clear Mem + Reste  -->  End" << "\n";
				}
			}
		}
	}

	file.open(Config.nameFile.stat, std::ofstream::out | std::ofstream::app);
	file << "Statistiques : \n";
	file << "				X min = " << Stat.Xmin << "\n";
	file << "				X max = " << Stat.Xmax << "\n";
	file << "				Y min = " << Stat.Ymin << "\n";
	file << "				Y max = " << Stat.Ymax << "\n";
	file << "				Z min = " << Stat.Zmin << "\n";
	file << "				Z max = " << Stat.Zmax << "\n";
	file << "				W min = " << Stat.Wmin << "\n";
	file << "				W max = " << Stat.Wmax << "\n";
	file << "				X step = " << Stat.Xstep << "\n";
	file << "				Y step = " << Stat.Ystep << "\n";
	file << "				Z step = " << Stat.Zstep << "\n";
	file << "				W step = " << Stat.Wstep << "\n";
	file << "				NbPoint plot = " << Stat.NbPoint << "\n";
	file.close();



	return 0;//Fin du programme
}