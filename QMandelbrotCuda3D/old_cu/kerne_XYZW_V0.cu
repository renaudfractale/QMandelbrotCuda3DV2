#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //lib W10
#include ""//lib W10
#include <iostream> // prompt Output
#include <fstream> //File Output
#include <math.h> //lib mayh
#include <stdio.h> // lib stantard
#include <hip/hip_fp16.h> // lib CUDA
#include <windows.h>
#include <fstream>
#include <string>
#define Dim_isFix 0;
#define Dim_end 20.0f;
#define Dim_start -20.0f;
#define Dim_NbStep 4;
#define Dim_step 10.0f;

#define ITER_MAX 255;
#define ITER_MIN 10;
#define ITER_isFix 0;

#define DEV 1;
#define FILTER 0;
#define POWER 2.0f;
#define ISSHOW 1;
#define RMAX 4.0f;

#define NBPOINTS 64;

// struct sur la gestion des dimensions
typedef struct 	struct_P_float {
	int isFix = Dim_isFix
	float start = Dim_start
	float end = Dim_end
	int NbStep = Dim_NbStep
	float step = Dim_step
} struct_P_float_T;

typedef struct 	struct_Iter {
	int max = ITER_MAX
	int min = ITER_MIN
	int isFix = ITER_isFix
} struct_Iter_T;

typedef struct 	struct_FileName {
	char root[100];
	char txt[110];
	char csv[110];
	char stat[110];
	char histo[110];
	char stl[110];
} struct_FileName_T;

// struct sur la gestion paramètres d'entré
typedef struct 	struct_P_All {
	struct_P_float_T X;
	struct_P_float_T Y;
	struct_P_float_T Z;
	struct_P_float_T W;
	struct_Iter_T Iter;
	int dev = DEV
	int filter=FILTER
	float power = POWER
	int isShow = ISSHOW
	struct_FileName_T nameFile;
	float rMax = RMAX
	int NbPointByStep = NBPOINTS
} struct_P_All_T;


int main(int argc, char *argv[])
{
	//Config
		struct_P_All_T Config;
		strcpy(Config.nameFile.root, "O");
	//Arg Help
		char Str_H[] = "-h";
		char Str_Help[] = "--help";
	
	//Arg X
		char Str_xFix[] = "-x";
		char Str_xMax[] = "-xmax";
		char Str_xMin[] = "-xmin";
		char Str_xNbStep[] = "-xNbStep";
	//Arg W
		char Str_wFix[] = "-w";
		char Str_wMax[] = "-wmax";
		char Str_wMin[] = "-wmin";
		char Str_wNbStep[] = "-wNbStep";
	//Arg Y
		char Str_yFix[] = "-y";
		char Str_yMax[] = "-ymax";
		char Str_yMin[] = "-ymin";
		char Str_yNbStep[] = "-yNbStep";
	//Arg Z
		char Str_zFix[] = "-z";
		char Str_zMax[] = "-zmax";
		char Str_zMin[] = "-zmin";
		char Str_zNbStep[] = "-zNbStep";
	//Arg Iter
		char Str_IterFix[] = "-iter";
		char Str_IterMax[] = "-iterMax";
		char Str_IterMin[] = "-iterMin";
	//Arg Dev
		char Str_dev[] = "-device";
	//Arg Filter
		char Str_filter[] = "-filter";
	//Arg Power
		char Str_power[] = "-power";
	//Arg IsShow
		char Str_IsShow[] = "-isShow";
	//Arg Output File
		char Str_Out[] = "-o";
	//Arg Rmax
		char Str_rMax[] = "-rMax";
	//Si il y a des Arguments
	if (argc > 1)
	{
		// dédection la commande -help pu -h
		for (int i = 1; i <= argc; i++)
		{
			if (strcmp(argv[i], Str_Help) == 0 || strcmp(argv[i], Str_H) == 0)
			{
				//Affiche Help
				FILE *fileman;
				char line[1000];
				fileman = fopen("man", "r");
				if (fileman != NULL)
				{
					while (std::fgets(line, 1000, fileman))
					{
						std::cout << line;
					}
					fclose(fileman);
					return 0; //Fin du programme
				}
				else
				{
					std::cout << "Error  : man no found" << "\n";
					return -1;
				}			
			}
		}
		std::cout << "¨Pass 1" << "\n";
		//Verification : si nb d'arguments est paire --> erreur
		if (argc % 2 == 0)
		{
			std::cout << "Error 00 : Argument impaire" << "\n";
			return -1;
		}
		else //: si nb d'arguments est impaire --> fonctionement normale
		{
			std::cout << "¨Pass 2" << "\n";
			for (int i = 1; i < argc; i += 2)
			{
				std::cout << "Analyse du couple d'arguments :  " << argv[i] << " " << argv[i + 1] << "\n";
				if (strcmp(argv[i], Str_xFix) == 0)
				{
					if (Config.X.isFix == 0) //Si premier config
					{
						Config.X.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xFix << ": value is not float " << "\n";
							return -1;
						}
						Config.X.start = value;
						Config.X.end = value;
						Config.X.NbStep = 1;
						Config.X.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_xFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wFix) == 0)
				{
					if (Config.W.isFix == 0) //Si premier config
					{
						Config.W.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wFix << ": value is not float " << "\n";
							return -1;
						}
						Config.W.start = value;
						Config.W.end = value;
						Config.W.NbStep = 1;
						Config.W.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_wFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yFix) == 0)
				{
					if (Config.Y.isFix == 0) //Si premier config
					{
						Config.Y.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yFix << ": value is not float " << "\n";
							return -1;
						}
						Config.Y.start = value;
						Config.Y.end = value;
						Config.Y.NbStep = 1;
						Config.Y.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_yFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zFix) == 0)
				{
					if (Config.Z.isFix == 0) //Si premier config
					{
						Config.Z.isFix = 1;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zFix << ": value is not float " << "\n";
							return -1;
						}
						Config.Z.start = value;
						Config.Z.end = value;
						Config.Z.NbStep = 1;
						Config.Z.step = 1.0f;
					}
					else
					{
						std::cout << "Error 02 " << Str_zFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xMax) == 0)
				{
					if (Config.X.isFix == 0 || Config.X.isFix == 2)
					{
						Config.X.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xMax << ": value is not float " << "\n";
							return -1;
						}
						Config.X.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wMax) == 0)
				{
					if (Config.W.isFix == 0 || Config.W.isFix == 2)
					{
						Config.W.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wMax << ": value is not float " << "\n";
							return -1;
						}
						Config.W.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yMax) == 0)
				{
					if (Config.Y.isFix == 0 || Config.Y.isFix == 2)
					{
						Config.Y.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yMax << ": value is not float " << "\n";
							return -1;
						}
						Config.Y.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zMax) == 0)
				{
					if (Config.Z.isFix == 0 || Config.Z.isFix == 2)
					{
						Config.Z.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zMax << ": value is not float " << "\n";
							return -1;
						}
						Config.Z.end = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_zMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xMin) == 0)
				{
					if (Config.X.isFix == 0 || Config.X.isFix == 2)
					{
						Config.X.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_xMin << ": value is not float " << "\n";
							return -1;
						}
						Config.X.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wMin) == 0)
				{
					if (Config.W.isFix == 0 || Config.W.isFix == 2)
					{
						Config.W.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_wMin << ": value is not float " << "\n";
							return -1;
						}
						Config.W.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yMin) == 0)
				{
					if (Config.Y.isFix == 0 || Config.Y.isFix == 2)
					{
						Config.Y.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_yMin << ": value is not float " << "\n";
							return -1;
						}
						Config.Y.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zMin) == 0)
				{
					if (Config.Z.isFix == 0 || Config.Z.isFix == 2)
					{
						Config.Z.isFix = 2;
						float value = (float)atof(argv[i + 1]);
						if (errno)
						{
							std::cout << "Error 01 " << Str_zMin << ": value is not float " << "\n";
							return -1;
						}
						Config.Z.start = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_zMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_xNbStep) == 0)
				{
					if (Config.X.isFix == 0 || Config.X.isFix == 2)
					{
						Config.X.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_xNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.X.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_xNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_wNbStep) == 0)
				{
					if (Config.W.isFix == 0 || Config.W.isFix == 2)
					{
						Config.W.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_wNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.W.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_wNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_yNbStep) == 0)
				{
					if (Config.Y.isFix == 0 || Config.Y.isFix == 2)
					{
						Config.Y.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_yNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.Y.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_zNbStep) == 0)
				{
					if (Config.Z.isFix == 0 || Config.Z.isFix == 2)
					{
						Config.Z.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_yNbStep << ": value is not int or value <= 0 " << "\n";
							return -1;
						}
						Config.Z.NbStep = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_yNbStep << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterFix) == 0)
				{
					if (Config.Iter.isFix == 0) //Si premier config
					{
						Config.Iter.isFix = 1;
						int value = atoi(argv[i + 1]);
						if (errno || value<=0)
						{
							std::cout << "Error 03 " << Str_IterFix << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config.Iter.max = value;
						Config.Iter.min = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterFix << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterMax) == 0)
				{
					if (Config.Iter.isFix == 0 || Config.Iter.isFix == 2) //Si premier config
					{
						Config.Iter.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_IterMax << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config.Iter.max = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterMax << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_IterMin) == 0)
				{
					if (Config.Iter.isFix == 0 || Config.Iter.isFix == 2) //Si premier config
					{
						Config.Iter.isFix = 2;
						int value = atoi(argv[i + 1]);
						if (errno || value <= 0)
						{
							std::cout << "Error 03 " << Str_IterMin << ": value is not int  or value <= 0" << "\n";
							return -1;
						}
						Config.Iter.min = value;
					}
					else
					{
						std::cout << "Error 02 " << Str_IterMin << ": value already assigned " << "\n";
						return -1;
					}
				}
				else if (strcmp(argv[i], Str_dev) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_dev << ": value is not int" << "\n";
						return -1;
					}
					Config.dev = value;
				}
				else if (strcmp(argv[i], Str_filter) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_filter << ": value is not int" << "\n";
						return -1;
					}
					Config.filter = value;
				}
				else if (strcmp(argv[i], Str_power) == 0)
				{
					float value = (float)atof(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_filter << ": value is not float" << "\n";
						return -1;
					}
					Config.power = value;
				}
				else if (strcmp(argv[i], Str_IsShow) == 0)
				{
					int value = atoi(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_IsShow << ": value is not int" << "\n";
						return -1;
					}
					Config.isShow = value;
				}
				else if (strcmp(argv[i], Str_rMax) == 0)
				{
					float value = (float)atof(argv[i + 1]);
					if (errno)
					{
						std::cout << "Error 01 " << Str_rMax << ": value is not float" << "\n";
						return -1;
					}
					Config.rMax = value;
				}
				else if (strcmp(argv[i], Str_Out) == 0)
				{
					if (strlen(argv[i + 1])<100)
						strcpy(Config.nameFile.root, argv[i + 1]);
					else
					{
						std::cout << "Error 04 strlen fileOutput must be inf to 100 signe \n";
						return -1;
					}
				}
				else
				{
					std::cout << "Warning 05  Arg not know : " << argv[i] << " " << argv[i + 1] << "\n";
				}
			}
		}
	}
	bool IsErrors = false;
	// Verification W
	if (Config.W.isFix == 1 && Config.W.start == Config.W.end)
	{
		;//OK
	}
	else if(Config.W.isFix == 2 || Config.W.isFix == 0)
	{
		Config.W.isFix = 2;
		if (Config.W.start < Config.W.end && Config.W.NbStep>1)
		{
			Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
		}
		else
		{
			std::cout << "Error W  :  wmax must be sup wmin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error W  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification X
	if (Config.X.isFix == 1 && Config.X.start == Config.X.end)
	{
		;//OK
	}
	else if (Config.X.isFix == 2 || Config.X.isFix == 0)
	{
		Config.X.isFix = 2; 
		if (Config.X.start < Config.X.end && Config.X.NbStep>1)
		{
			Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
		}
		else
		{
			std::cout << "Error X  :  xmax must be sup xmin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error X  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification Y
	if (Config.Y.isFix == 1 && Config.Y.start == Config.Y.end)
	{
		;//OK
	}
	else if (Config.Y.isFix == 2 || Config.Y.isFix == 0)
	{
		Config.Y.isFix = 2; 
		if (Config.Y.start < Config.Y.end && Config.Y.NbStep>1)
		{
			Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
		}
		else
		{
			std::cout << "Error Y  :  ymax must be sup ymin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error Y  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification Z
	if (Config.Z.isFix == 1 && Config.Z.start == Config.Z.end)
	{
		;//OK
	}
	else if (Config.Z.isFix == 2 || Config.Z.isFix == 0)
	{
		Config.Z.isFix = 2; 
		if (Config.Z.start < Config.Z.end && Config.Z.NbStep>1)
		{
			Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
		}
		else
		{
			std::cout << "Error Z  :  zmax must be sup zmin AND nbStep must be sup 1\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error Z  :  Error unknow\n";
		IsErrors=true;
	}
	// Verification Iter
	if (Config.Iter.isFix == 1)
	{
		if (Config.Iter.min == Config.Iter.max && Config.Iter.max > 0)
		{
			; //OK
		}
		else
		{
			std::cout << "Error Iter  :  value must be sup at 0\n";
			IsErrors=true;
		}
	}
	else if (Config.Iter.isFix == 2 || Config.Iter.isFix == 0)
	{
		Config.Iter.isFix = 2;
		if (Config.Iter.min < Config.Iter.max)
		{
			; // OK
		}
		else
		{
			std::cout << "Error Iter  :  max must be sup min\n";
			IsErrors=true;
		}
	}
	else
	{
		std::cout << "Error Iter  :  Error unknow\n";
		IsErrors=true;
	}
	// Vérification dev
	int count;
	hipGetDeviceCount(&count);
	if (Config.dev >= 0 && Config.dev < count)
	{
		; //OK
	}
	else
	{
		std::cout << "Error dev :  dev must be between 0 and "<< count -1 <<"\n";
		IsErrors=true;
	}
	//Verification filter
	if (Config.filter >= 0 && Config.filter <= 2)
	{
		; //OK
	}
	else
	{
		std::cout << "Error filter :  filter must be between 0 and 2\n";
		IsErrors=true;
	}
	//Verification power
	if (Config.power >= 2.0f && Config.power <= 50.0f)
	{
		; //OK
	}
	else
	{
		std::cout << "Error power :  power must be between 2.0 and 50.0\n";
		IsErrors=true;
	}
	//Verification IsShow
	if (Config.isShow == 0 || Config.isShow == 1)
	{
		; //OK
	}
	else
	{
		std::cout << "Error isShow :  isShow must be between 0 and 1 \n";
		IsErrors=true;
	}
	//Verification rMax
	if (Config.rMax >0.0f)
	{
		; //OK
	}
	else
	{
		std::cout << "Error rMax :  rMax must be sup 0.0 \n";
		IsErrors = true;
	}
	if (IsErrors)
		return -1;

	// creation des fichiers :
	strcpy(Config.nameFile.csv, Config.nameFile.root);
	strcpy(Config.nameFile.histo, Config.nameFile.root);
	strcpy(Config.nameFile.stat, Config.nameFile.root);
	strcpy(Config.nameFile.stl, Config.nameFile.root);
	strcpy(Config.nameFile.txt, Config.nameFile.root);
	
	strcat(Config.nameFile.csv, ".csv");
	strcat(Config.nameFile.histo, ".histo");
	strcat(Config.nameFile.stat, ".stat");
	strcat(Config.nameFile.stl, ".stl");
	strcat(Config.nameFile.txt, ".txt");

	//Affichage de la config
	std::ofstream file;
	file.open(Config.nameFile.stat);
	file << "Parameters Current : " << "\n";
	file << "				W_start = " << Config.W.start << ", W_end = " << Config.W.end << ", W_Step = " << Config.W.step << ", W_NbStep = " << Config.W.NbStep << "\n";
	file << "				X_start = " << Config.X.start << ", X_end = " << Config.X.end << ", X_Step = " << Config.X.step << ", X_NbStep = " << Config.X.NbStep << "\n";
	file << "				Y_start = " << Config.Y.start << ", Y_end = " << Config.Y.end << ", Y_Step = " << Config.Y.step << ", Y_NbStep = " << Config.Y.NbStep << "\n";
	file << "				Z_start = " << Config.Z.start << ", Z_end = " << Config.Z.end << ", Z_Step = " << Config.Z.step << ", Z_NbStep = " << Config.Z.NbStep << "\n";
	file << "				Root FileOutput = " << Config.nameFile.root << "\n";
	file << "				iterMax = " << Config.Iter.max << "\n";
	file << "				iterMin = " << Config.Iter.min << "\n";
	file << "				rMax = " << Config.rMax << "\n";
	file << "				Filter = " << Config.filter<< "\n";
	file << "				Power = " << Config.power << "\n";
	file << "				dev = " << Config.dev << "\n";
	file << "				IsShow = " << Config.isShow << "\n";
	file << "				NbPoints per step = " << Config.NbPointByStep << "\n";
	file << "				ouput File :  " << Config.nameFile.root << "\n";
	file << "cmd for use this configuration: " << "\n";
	file << "               " << argv[0] << "  ";
	if (Config.W.isFix == 1)
		file << Str_wFix << " " << Config.W.start;
	else
	{
		file << Str_wMin << " " << Config.W.start << " ";
		file << Str_wMax << " " << Config.W.end << " ";
		file << Str_wNbStep << " " << Config.W.NbStep << " ";
	}
	if (Config.X.isFix == 1)
		file << Str_xFix << " " << Config.X.start;
	else
	{
		file << Str_xMin << " " << Config.X.start << " ";
		file << Str_xMax << " " << Config.X.end << " ";
		file << Str_xNbStep << " " << Config.X.NbStep << " ";
	}
	if (Config.Y.isFix == 1)
		file << Str_yFix << " " << Config.Y.start;
	else
	{
		file << Str_yMin << " " << Config.Y.start << " ";
		file << Str_yMax << " " << Config.Y.end << " ";
		file << Str_yNbStep << " " << Config.Y.NbStep << " ";
	}
	if (Config.Z.isFix == 1)
		file << Str_zFix << " " << Config.Z.start;
	else
	{
		file << Str_zMin << " " << Config.Z.start << " ";
		file << Str_zMax << " " << Config.Z.end << " ";
		file << Str_zNbStep << " " << Config.Z.NbStep << " ";
	}
	if (Config.Iter.isFix == 1)
		file << Str_IterFix << " " << Config.Iter.max;
	else
	{
		file << Str_IterMin << " " << Config.Iter.min << " ";
		file << Str_IterMax << " " << Config.Iter.max << " ";
	}
	file << Str_dev << " " << Config.dev << " ";
	file << Str_filter << " " << Config.filter << " ";
	file << Str_power << " " << Config.power << " ";
	file << Str_IsShow << " " << Config.isShow << " ";
	file << Str_rMax << " " << Config.rMax << " ";
	file << Str_Out << " " << Config.nameFile.root << " ";

	file << "\n";
	file.close();

	//Affiche Help
	FILE *fileman;
	char line[1000];
	fileman = fopen(Config.nameFile.stat, "r");
	if (fileman != NULL)
	{
		while (std::fgets(line, 1000, fileman))
		{
			std::cout << line;
		}
		fclose(fileman);
	}
	else
		std::cout << "Error  : "<< Config.nameFile.stat <<" no found" << "\n";
	return 0;//Fin du programme
}