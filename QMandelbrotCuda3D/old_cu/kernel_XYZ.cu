#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //lib W10
#include ""//lib W10
#include <iostream> // prompt Output
#include <fstream> //File Output
#include <math.h> //lib mayh
#include <stdio.h> // lib stantard
#include <hip/hip_fp16.h> // lib CUDA
#include <windows.h>
#define NbPointPerStep 64;
// Pour X,Y,Z et W
typedef struct 	struct_P_float {
	float start;
	float end;
	int NbPoints;
	float step;
} struct_P_float_T;

typedef struct 	struct_Stat_float {
	float Wmin;
	float Wmax;
	float Xmin;
	float Xmax;
	float Ymin;
	float Ymax;
	float Zmin;
	float Zmax;
	unsigned long NbPoint;
} struct_Stat_float_T;

typedef struct 	struct_P_Power {
	float value;
} struct_P_Power_T;

typedef struct 	struct_P_Iter {
	int start;
	int end;
} struct_P_Iter_T;


typedef struct 	struct_P_Rlimit {
	float value;
} struct_P_Rlimit_T;

typedef struct 	struct_P_Simulation {
	//Quaternions
	struct_P_float_T X;
	struct_P_float_T Y;
	struct_P_float_T Z;
	struct_P_float_T W;
	//Parametre Fixe
	struct_P_Iter_T Iter;
	float Rlimit;
	//Parametrer variable systematique
	float Power;
	int Filter;
} struct_P_Simulation_T;

typedef struct 	struct_Q {
	float x;
	float y;
	float z;
	float w;
} struct_Q_T;

struct_P_Simulation_T *P_Simulation_DEVICE;
short *Tab_Iter;
__host__  void CreateQ_By_float_H(struct_Q_T *out, float x, float y, float z, float w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

__host__  float  Get_QNorm_H(struct_Q_T *Q)
{
	return sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
}

__host__ void Get_QPow_H(struct_Q_T *Q, float pow)
{
	float A = Get_QNorm_H(Q);
	float theta = 0.0f;
	float B = 0.0f;
	float R = 0.0f;
	if (pow > 0.0f && A>0.000001f)
	{
		float coef = 1.0f;
		if (A<1.0f)
		{
			//printf("%f *******\n", A);
			coef = 1 / A;
			Q->x *= coef;
			Q->y *= coef;
			Q->z *= coef;
			Q->z *= coef;

		}
		A = Get_QNorm_H(Q);
		//printf("%f +++++++++\n", A);
		theta = acosf(Q->w / A)*pow;
		B = sqrt(A*A - Q->w*Q->w);
		R = exp2f(logf(A / coef)* pow);
		Q->x = R*sinf(theta)*(Q->x / B);
		Q->y = R*sinf(theta)*(Q->y / B);
		Q->z = R*sinf(theta)*(Q->z / B);
		Q->z = R*cosf(theta);

	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0f;
		Q->x = 0.0f;
		Q->y = 0.0f;
		Q->z = 0.0f;

	}
}

__host__ int  GetQIter_H(struct_P_Simulation_T *P_Simulation_DEVICE, int  *x_filter, int  *y_filter, int *z_filter, int *w_filter)
{
	//int Tempindex = 0;
	struct_Q_T Q_Current;
	float w, x, y, z;
	int iter_computed;
	//X
	x = ((float)*x_filter)*P_Simulation_DEVICE->X.step + P_Simulation_DEVICE->X.start;
	//Y
	y = ((float)*y_filter)*P_Simulation_DEVICE->Y.step + P_Simulation_DEVICE->Y.start;
	//Z
	z = ((float)*z_filter)*P_Simulation_DEVICE->Z.step + P_Simulation_DEVICE->Z.start;
	//W
	w = ((float)*w_filter)*P_Simulation_DEVICE->W.step + P_Simulation_DEVICE->W.start;



	CreateQ_By_float_H(&Q_Current, x, y, z, w);

	for (iter_computed = 0; iter_computed <= P_Simulation_DEVICE->Iter.end; iter_computed++)
	{
		Get_QPow_H(&Q_Current, P_Simulation_DEVICE->Power);
		Q_Current.x += x;
		Q_Current.y += y;
		Q_Current.z += z;
		Q_Current.w += w;

		if (Get_QNorm_H(&Q_Current) > P_Simulation_DEVICE->Rlimit)
		{
			if (iter_computed > 0)
				iter_computed--;
			return iter_computed;
		}
	}
	if (iter_computed > 0)
		iter_computed--;
	return iter_computed;
}
__host__ bool  FilterQ_H(int *Filter, int *Nx, int *Ny, int *Nz, int *Nw, int iter, struct_P_Simulation_T *P_Simulation_DEVICE)
{
	if (*Filter == 0)
		return true;
	int w_filter = 0;
	int iter_computed = 0;
	for (int x_filter = *Nx - 1; x_filter <= *Nx + 1; x_filter++)
	{
		for (int y_filter = *Ny - 1; y_filter <= *Ny + 1; y_filter++)
		{
			for (int z_filter = *Nz - 1; z_filter <= *Nz + 1; z_filter++)
			{
				iter_computed = GetQIter_H(P_Simulation_DEVICE, &x_filter, &y_filter, &z_filter, &w_filter);
				if (*Filter == 1)
				{
					if (iter_computed != iter)
						return true;
				}
				else //filter==2
				{
					if (iter_computed == 0)
						return true;
				}
			}
		}
	}
	return false;
}

__device__  void CreateQ_By_float(struct_Q_T *out, float x, float y, float z, float w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

__device__  float  Get_QNorm(struct_Q_T *Q)
{
	return sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
}

__device__ void Get_QPow(struct_Q_T *Q, float pow)
{
	float A = Get_QNorm(Q);
	float theta = 0.0f;
	float B = 0.0f;
	float R = 0.0f;
	if (pow > 0.0f && A>0.000001f)
	{
		float coef = 1.0f;
		if (A<1.0f)
		{
			//printf("%f *******\n", A);
			coef = 1 / A;
			Q->x *= coef;
			Q->y *= coef;
			Q->z *= coef;
			Q->z *= coef;

		}
		A = Get_QNorm(Q);
		//printf("%f +++++++++\n", A);
		theta = acosf(Q->w / A)*pow;
		B = sqrt(A*A - Q->w*Q->w);
		R = exp2f(logf(A / coef)* pow);
		Q->x = R*sinf(theta)*(Q->x / B);
		Q->y = R*sinf(theta)*(Q->y / B);
		Q->z = R*sinf(theta)*(Q->z / B);
		Q->z = R*cosf(theta);

	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0f;
		Q->x = 0.0f;
		Q->y = 0.0f;
		Q->z = 0.0f;

	}
}

//
//__device__  void  Get_QAdd(struct_Q_T *Q1, struct_Q_T *Q2)
//{
//	Q1->x += Q2->x;
//	Q1->y += Q2->y;
//	Q1->z += Q2->z;
//	Q1->w += Q2->w;
//}

//__device__ void GetIterMax(int *iter, float *x, float *y, float *z, float *w, const struct_P_Simulation_T *P_Simulation)
//{
//	struct_Q_T Q_Current; 
//	CreateQ_By_float(&Q_Current, *x, *y, *z, *w);
//
//	for (iter = 0; *iter <= P_Simulation->Iter.end; iter++)
//	{
//		Get_QPow(&Q_Current, P_Simulation->Power);
//		Q_Current.x += *x;
//		Q_Current.y += *y;
//		Q_Current.z += *z;
//		Q_Current.w += *w;
//
//		if (Get_QNorm(&Q_Current) > P_Simulation->Rlimit)
//			goto Fin;
//	}
//Fin:
//	if (iter > 0)
//		iter--;
//}


// CUDA kernel to Compute itermax of quaternion
__global__ void kernel(const struct_P_Simulation_T *P_Simulation, short *Tab_Iter)
{
	//int Tempindex = 0;
	struct_Q_T Q_Current;
	float w, x, y, z;
	int iter = 0;
	//X
	x = ((float)blockIdx.y)*P_Simulation->X.step + P_Simulation->X.start;
	//Y
	y = ((float)blockIdx.x)*P_Simulation->Y.step + P_Simulation->Y.start;
	//Z
	z = ((float)threadIdx.x)*P_Simulation->Z.step + P_Simulation->Z.start;
	//W
	w = ((float)threadIdx.y)*P_Simulation->W.step + P_Simulation->W.start;

	CreateQ_By_float(&Q_Current, x, y, z, w);

	for (iter = 0; iter <= P_Simulation->Iter.end; iter++)
	{
		Get_QPow(&Q_Current, P_Simulation->Power);
		Q_Current.x += x;
		Q_Current.y += y;
		Q_Current.z += z;
		Q_Current.w += w;

		if (Get_QNorm(&Q_Current) > P_Simulation->Rlimit)
			goto Fin;
	}
Fin:
	if (iter > 0)
		iter--;
	Tab_Iter[(blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x] = (short)iter;
}
//
//
//// CUDA kernel to Compute itermax of quaternion
//__global__ void kernel(const struct_P_Simulation_T *P_Simulation, short *Tab_Iter)
//{
//	struct_Q_T Q_Current;
//	float w, y, z;
//	int iter = 0;
//	//Y
//	y = ((float)blockIdx.y)*P_Simulation->Y.step + P_Simulation->Y.start;
//	//Z
//	z = ((float)blockIdx.x)*P_Simulation->Z.step + P_Simulation->Z.start;
//	//W
//	w = ((float)threadIdx.x)*P_Simulation->W.step + P_Simulation->W.start;
//
//	CreateQ_By_float(&Q_Current, P_Simulation->X.value, y, z, w);
//
//	for (iter = 0; iter <= P_Simulation->Iter.end; iter++)
//	{
//		Get_QPow(&Q_Current, P_Simulation->Power);
//		Q_Current.x += P_Simulation->X.value;
//		Q_Current.y += y;
//		Q_Current.z += z;
//		Q_Current.w += w;
//
//		if (Get_QNorm(&Q_Current) > P_Simulation->Rlimit)
//			goto Fin;
//	}
//	Fin:
//	if (iter > 0)
//		iter--;
//	Tab_Iter[(blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x] = (short)iter;
//}

int main(int argc, char *argv[])
{

	//#################### CONFIG par DEFAUT #################
	struct_P_float_T ParameterDelaults;
	// ******************** NbPoints **************
	ParameterDelaults.NbPoints = 8;
	char Str_NbPoints[] = "-NbPoints";
	//********************* Parameter y,z,w *************
	ParameterDelaults.start = -3.3f;
	char Str_Q_start[] = "-Q_start";
	ParameterDelaults.end = 2.5f;
	char Str_Q_end[] = "-Q_end";
	ParameterDelaults.step = (ParameterDelaults.end - ParameterDelaults.start) / ((float)ParameterDelaults.NbPoints - 1);
	//char Str_Q_step[] = "-Q_step";
	//*********************  NameFile Output *************
	char NameFile[110];
	char NameFile_csv[110];
	char NameFile_histo[110];
	char NameFile_stat[110];
	char NameFile_txt[110];
	strcpy(NameFile, "OutputFile");
	char Str_NameFile[] = "-o";
	//*********************  X value *************
	float Parameter_Fix = 0.3375f;
	char Str_X[] = "-Fix";

	float POWER = 3.5;
	char Str_Power[] = "-Power";

	char Str_H[] = "-h";
	char Str_Help[] = "--help";

	int itermax = 255;
	float Rmax = 4.0;

	int dev = 0;
	char Str_dev[] = "-dev";

	bool IsShow = false;
	char Str_IsShow[] = "-IsShow";

	int Filter = 2;
	char Str_Filter[] = "-Filter";
	if (argc > 1)
	{
		if (argc % 2 == 0)
		{
			for (int i = 1; i <= argc; i++)
			{
				if (strcmp(argv[i], Str_Help) == 0 || strcmp(argv[i], Str_H) == 0)
				{
					int count;
					hipGetDeviceCount(&count);

					std::cout << "Help :  \n";
					std::cout << "      -NbPoints : numbers of points \n";
					std::cout << "                  Ctrt 01 : if start = end , NbPoints must be equal to 1 \n";
					std::cout << "                  Ctrt 02 : if start < end , NbPoints must be sup to 1 \n";
					std::cout << "                  Ctrt 03 : NbPoints must be type int\n";
					std::cout << "      -Q_start : value  of start \n";
					std::cout << "                  Ctrt 03 :  start >= end \n";
					std::cout << "                  Ctrt 03 : start must be type float\n";
					std::cout << "      -Q_end : value  of end \n";
					std::cout << "                  Ctrt 03 :  start >= end \n";
					std::cout << "                  Ctrt 03 : end must be type float\n";
					std::cout << "      -o : Output File \n";
					std::cout << "                  Ctrt 05 :  len  must be inf 100 char \n";
					std::cout << "      -X : value of x \n";
					std::cout << "                  Ctrt 03 : x must be type float\n";
					std::cout << "      -Power : value of Power \n";
					std::cout << "                  Ctrt 03 : Power must be type float\n";
					std::cout << "                  Ctrt 03 : Power must be sup 0.0 \n";
					std::cout << "      -dev : index of device GPU, 0 by default \n";
					std::cout << "            --> you have " << count << " device(s) in your PC \n";
					std::cout << "                  Ctrt 03 : dev must be type int\n";
					std::cout << "                  Ctrt 03 : dev must be sup or equal 0 \n";
					std::cout << "      -IsShow : bool if show message in cmd, true (1) by default \n";
					std::cout << "                  Ctrt 03 : IsShow must be type int (0 or 1)\n";
					std::cout << "      -Filter : int if you would to filter clould points, 0 by default \n";
					std::cout << "                  Ctrt 03 : Filter must be type int (0 or 1 or 2)\n";
					std::cout << "                  Option 0 : Not Filter \n";
					std::cout << "                  Option 1 : Filter on iter_around  must be equal inter_computed \n";
					std::cout << "                  Option 2 : Filter on iter_around  must be sup 0 \n";
					std::cout << "      -h / --help : show help \n";
					std::cout << "      Example :\n";
					std::cout << "               Programme.exe -X 0.3375 -Q_start -3.0 -Q_end 3.0 -NbPoints 4 -Power 2.5 -o FileOutput \n";
					std::cout << "               Programme.exe -X 0.3375 -Q_start -4.0 -Q_end 4.0 -NbPoints 4 -o FileOutput2 -IsShow 0 -dev 1 -Power 2.0\n";
					std::cout << "      Version : 0.15 du 07 Juillet 2018\n";
					std::cout << "      Auteur : Renaud HENRY\n";
					std::cout << "      siteweb : http://fractale.io/ \n";
					return 0;
				}
			}
			std::cout << "Error 00 : Argument impaire" << "\n";
			return -1;
		}
		for (int i = 1; i < argc; i += 2)
		{
			std::cout << "Analyse du couple d'arguments :  " << argv[i] << " " << argv[i + 1] << "\n";
			if (strcmp(argv[i], Str_Q_start) == 0)
			{
				ParameterDelaults.start = (float)atof(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 02 " << Str_Q_start << ": value is not float " << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_Q_end) == 0))
			{
				ParameterDelaults.end = (float)atof(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 02 " << Str_Q_end << ": value is not type float " << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_X) == 0))
			{
				Parameter_Fix = (float)atof(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 02 " << Str_X << ": value is not type float " << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_NbPoints) == 0))
			{
				ParameterDelaults.NbPoints = atoi(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 02 " << Str_NbPoints << ": value is not type int " << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_Power) == 0))
			{
				POWER = (float)atof(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 09 " << Str_Power << ": value is not type float " << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_IsShow) == 0))
			{
				int temp = atoi(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 12 " << Str_IsShow << ": value is not type int " << "\n";
					return -1;
				}
				if (temp == 0)
					IsShow = false;
				else if (temp == 1)
					IsShow = true;
				else
				{
					std::cout << "Error 13 " << Str_IsShow << ": must be equal at 1 ou 0" << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_dev) == 0))
			{
				dev = atoi(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 11 " << Str_dev << ": value is not type int " << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_Filter) == 0))
			{
				Filter = atoi(argv[i + 1]);
				if (errno)
				{
					std::cout << "Error 14 " << Str_Filter << ": value is not type int " << "\n";
					return -1;
				}
			}
			else if ((strcmp(argv[i], Str_NameFile) == 0))
			{
				if (strlen(argv[i + 1])<100)
					strcpy(NameFile, argv[i + 1]);
				else
				{
					std::cout << "Error 07 strlen fileOutput must be inf to 100 signe \n";
					return -1;
				}
			}
			else
			{
				std::cout << "Warning 08  Arg not know : " << argv[i] << " " << argv[i + 1] << "\n";
			}
		}
	}
	if (POWER <= 0.0f)
	{
		std::cout << "Error 10 POWER < 0.0f : " << POWER << "<" << 0.0f << "\n";
		return -1;
	}
	if (ParameterDelaults.end < ParameterDelaults.start)
	{
		std::cout << "Error 03 end < start : " << ParameterDelaults.end << "<" << ParameterDelaults.start << "\n";
		return -1;
	}
	if (ParameterDelaults.end == ParameterDelaults.start && ParameterDelaults.NbPoints != 1)
	{
		std::cout << "Warning  04 end == start  and NbPoints !=1:  So NbPoints force to 1\n";
		ParameterDelaults.NbPoints = 1;
	}
	if (ParameterDelaults.NbPoints > 1)
	{
		ParameterDelaults.step = (ParameterDelaults.end - ParameterDelaults.start) / ((float)ParameterDelaults.NbPoints);
		if (ParameterDelaults.step < 0.0001f)
		{
			std::cout << "Error 05 step < 0.0001 : " << ParameterDelaults.end << "<" << 0.0001f << "\n";
			std::cout << "step must be sup at 0.0001 \n";
			return -1;
		}
	}
	else if (ParameterDelaults.end != ParameterDelaults.start)
	{
		std::cout << "Error 06 NbPoints < 1:  So NbPoints must be sup 0 \n";
		return -1;
	}
	if (Filter >3)
	{
		std::cout << "Error 15 " << Str_Filter << ": value of dev > 3, " << Filter << " > " << 3 << "\n";
		return -1;
	}
	if (Filter < 0)
	{
		std::cout << "Error 16 " << Str_Filter << ": value of dev < 0 , " << Filter << " < " << 0 << "\n";
		return -1;
	}
	int count;
	hipGetDeviceCount(&count);
	if (count < dev)
	{
		std::cout << "Error 12 " << Str_dev << ": value of dev > countdevice, " << dev << " > " << count << "\n";
		return -1;
	}
	if (dev < 0)
	{
		std::cout << "Error 13 " << Str_dev << ": value of dev < 0 , " << dev << " <" << 0 << "\n";
		return -1;
	}
	//CST 
	const int NbPoints = NbPointPerStep;

	/****** Print Parmaters Used ******/
	std::cout << "Parameters Current : " << "\n";
	std::cout << "		Q_start = " << ParameterDelaults.start << ", Q_end = " << ParameterDelaults.end << ", Q_Step = " << ParameterDelaults.step << ", Nbpoints = " << ParameterDelaults.NbPoints << "\n";
	std::cout << "		Fix = " << Parameter_Fix << "\n";
	std::cout << "		FileOutput = " << NameFile << "\n";
	std::cout << "		itermax = " << itermax << "\n";
	std::cout << "		Rmax = " << Rmax << "\n";
	std::cout << "		Filter = " << Filter << "\n";
	std::cout << "		POWER = " << POWER << "\n";
	std::cout << "		dev = " << dev << "\n";
	std::cout << "		IsShow = " << IsShow << "\n";
	std::cout << "				NpStep =  " << ParameterDelaults.NbPoints << "\n";
	std::cout << "				NbPoints per step = " << NbPoints << "\n";
	std::cout << "				ouput File :  " << NameFile << "\n";
	std::cout << "cmd for use this configuration: " << "\n";
	std::cout << "               Programme.exe -Fix " << Parameter_Fix << " -Q_start " << ParameterDelaults.start << " -Q_end " << ParameterDelaults.end << " -NbPoints " << ParameterDelaults.NbPoints << " -o " << NameFile << " -IsShow " << IsShow << " -dev " << dev << " -Power " << POWER << " -Filter " << Filter << " \n";




	std::ofstream file;
	strcpy(NameFile_stat, NameFile);
	strcat(NameFile_stat, ".stat");
	file.open(NameFile_stat);
	file << "Parameters Current : " << "\n";
	file << "				Q_start = " << ParameterDelaults.start << ", Q_end = " << ParameterDelaults.end << ", Q_Step = " << ParameterDelaults.step << ", Nbpoints = " << ParameterDelaults.NbPoints << "\n";
	file << "				Fix = " << Parameter_Fix << "\n";
	file << "				FileOutput = " << NameFile << "\n";
	file << "				itermax = " << itermax << "\n";
	file << "				Rmax = " << Rmax << "\n";
	file << "				Filter = " << Filter << "\n";
	file << "				POWER = " << POWER << "\n";
	file << "				dev = " << dev << "\n";
	file << "				IsShow = " << IsShow << "\n";
	file << "				NpStep =  " << ParameterDelaults.NbPoints << "\n";
	file << "				NbPoints per step = " << NbPoints << "\n";
	file << "				ouput File :  " << NameFile << "\n";
	file << "cmd for use this configuration: " << "\n";
	file << "               Programme.exe -Fix " << Parameter_Fix << " -Q_start " << ParameterDelaults.start << " -Q_end " << ParameterDelaults.end << " -NbPoints " << ParameterDelaults.NbPoints << " -o " << NameFile << " -IsShow " << IsShow << " -dev " << dev << " -Power " << POWER << " -Filter " << Filter << " \n";
	file.close();

	//Init Stat
	struct_Stat_float_T Stat;
	Stat.Xmax = ParameterDelaults.start;
	Stat.Xmin = ParameterDelaults.end;

	Stat.Ymax = ParameterDelaults.start;
	Stat.Ymin = ParameterDelaults.end;

	Stat.Zmax = ParameterDelaults.start;
	Stat.Zmin = ParameterDelaults.end;

	Stat.Wmax = ParameterDelaults.start;
	Stat.Wmin = ParameterDelaults.end;

	Stat.NbPoint = (unsigned long)0;
	/********  Clear File ************/
	std::ofstream filetxt;
	strcpy(NameFile_txt, NameFile);
	strcat(NameFile_txt, ".txt");
	filetxt.open(NameFile_txt);
	filetxt.close();


	strcpy(NameFile_csv, NameFile);
	strcat(NameFile_csv, ".csv");
	file.open(NameFile_csv);
	file << "X;Y;Z;W;iter;\n";
	file.close();

	strcpy(NameFile_histo, NameFile);
	strcat(NameFile_histo, ".histo");
	file.open(NameFile_histo);
	file << "index;";
	for (int i = 0; i <= itermax; i++)
		file << i << ";";
	file << "\n";
	file.close();

	//#################### Constante(s) #################

	const int  maxMaster = ParameterDelaults.NbPoints * ParameterDelaults.NbPoints  * ParameterDelaults.NbPoints;
	const int  maxMinor = NbPoints * NbPoints  * NbPoints;

	//#################### Variables(s) #################

	int X = 0;
	int Y = 0;
	int Z = 0;

	int Tab_Histo[300];
	int  Nbpoint_iter = 0;

	hipSetDevice(dev);
	for (int index = 0; index < maxMaster; index++)
	{
		int indexTemp = index;

		X = indexTemp / (ParameterDelaults.NbPoints * ParameterDelaults.NbPoints);
		indexTemp = indexTemp - (ParameterDelaults.NbPoints * ParameterDelaults.NbPoints)*X;

		Y = indexTemp / (ParameterDelaults.NbPoints);
		indexTemp = indexTemp - (ParameterDelaults.NbPoints * Y);

		Z = indexTemp;


		if (IsShow)
			std::cout << "hipMallocManaged Config  -->  Start" << "\n";
		// Allocate Unified Memory -- accessible from CPU or GPU
		hipMallocManaged(&P_Simulation_DEVICE, sizeof(struct_P_Simulation_T));
		hipMallocManaged(&Tab_Iter, maxMinor * sizeof(short));
		if (IsShow)
			std::cout << "hipMallocManaged Config  -->  End " << "\n";
		if (IsShow)
			std::cout << "P_Simulation Config  -->  Start" << "\n";
		//Parametrage Iter
		P_Simulation_DEVICE->Iter.end = itermax;
		P_Simulation_DEVICE->Iter.start = 10;

		//Parametrage Power
		P_Simulation_DEVICE->Power = POWER;

		//Parametrage Rmax
		P_Simulation_DEVICE->Rlimit = Rmax;

		//Parametrage X
		P_Simulation_DEVICE->X.start = (float)X*ParameterDelaults.step + ParameterDelaults.start;
		P_Simulation_DEVICE->X.end = (float)(X + 1)*ParameterDelaults.step + ParameterDelaults.start;
		P_Simulation_DEVICE->X.NbPoints = NbPoints;
		P_Simulation_DEVICE->X.step = (P_Simulation_DEVICE->X.end - P_Simulation_DEVICE->X.start) / (P_Simulation_DEVICE->X.NbPoints - 1);

		//Parametrage Y
		P_Simulation_DEVICE->Y.start = (float)Y*ParameterDelaults.step + ParameterDelaults.start;
		P_Simulation_DEVICE->Y.end = (float)(Y + 1)*ParameterDelaults.step + ParameterDelaults.start;
		P_Simulation_DEVICE->Y.NbPoints = NbPoints;
		P_Simulation_DEVICE->Y.step = (P_Simulation_DEVICE->Y.end - P_Simulation_DEVICE->Y.start) / (P_Simulation_DEVICE->Y.NbPoints - 1);

		//Parametrage Z
		P_Simulation_DEVICE->Z.start = (float)Z*ParameterDelaults.step + ParameterDelaults.start;
		P_Simulation_DEVICE->Z.end = (float)(Z + 1)*ParameterDelaults.step + ParameterDelaults.start;
		P_Simulation_DEVICE->Z.NbPoints = NbPoints;
		P_Simulation_DEVICE->Z.step = (P_Simulation_DEVICE->Z.end - P_Simulation_DEVICE->Z.start) / (P_Simulation_DEVICE->Z.NbPoints - 1);

		//Parametrage W
		P_Simulation_DEVICE->W.start = Parameter_Fix;
		P_Simulation_DEVICE->W.end = Parameter_Fix;
		P_Simulation_DEVICE->W.NbPoints = 1;
		P_Simulation_DEVICE->W.step = 0;
		if (IsShow)
			std::cout << "P_Simulation Config  -->  End" << "\n";

		std::cout << "P_Simulation Config No " << index + 1 << " sur  " << maxMaster << "\n";

		if (IsShow)
			std::cout << "Tab_Iter and Tab_Histo Init  -->  Start" << "\n";
		for (int i = 0; i < maxMinor; i++) {
			Tab_Iter[i] = (short)0;

		}

		for (int i = 0; i <= itermax; i++)
			Tab_Histo[i] = 0;
		if (IsShow)
			std::cout << "Tab_Iter and Tab_Histo Init -->  End" << "\n";

		if (IsShow)
			std::cout << "Compude GPU -->  Start" << "\n";
		int NbThreadPerBlock = NbPoints;
		int NbBlockPerGrid = NbPoints;
		dim3 grid(NbBlockPerGrid, NbBlockPerGrid, 1);
		dim3 block(NbThreadPerBlock, 1, 1);
		kernel << <grid, block >> >(P_Simulation_DEVICE, Tab_Iter);
		if (IsShow)
			std::cout << "Compude GPU -->  End" << "\n";

		if (IsShow)
			std::cout << "hipDeviceSynchronize-->  Start" << "\n";
		hipDeviceSynchronize();
		if (IsShow)
			std::cout << "hipDeviceSynchronize -->  End" << "\n";

		if (IsShow)
			std::cout << "Analyzer Simulation -->  Start" << "\n";
		Nbpoint_iter = 0;
		for (int i = 0; i < maxMinor; i++)
		{
			if (Tab_Iter[i] > 0)
				Nbpoint_iter++;
			Tab_Histo[Tab_Iter[i]]++;
		}
		if (IsShow)
			std::cout << "Nb point Nbpoint_iter = " << Nbpoint_iter << "\n";
		if (IsShow)
			std::cout << "Soit  :  " << (float)(Nbpoint_iter / (maxMinor / 10000)) / 100.0f << "%  soit " << Nbpoint_iter << "pt sur " << maxMinor << "pt \n";
		if (IsShow)
			std::cout << "Analyzer Simulation -->  End" << "\n";

		if (IsShow)
			std::cout << "Write Histogram -->  Start" << "\n";
		file.open(NameFile_histo, std::ofstream::out | std::ofstream::app);
		file << index << ";";
		for (int i = 0; i <= itermax; i++)
			file << Tab_Histo[i] << ";";
		file << "\n";
		file.close();
		if (IsShow)
			std::cout << "Write Histogram -->  End" << "\n";

		if (IsShow)
			std::cout << "Write csv -->  Start" << "\n";
		file.open(NameFile_csv, std::ofstream::out | std::ofstream::app);
		filetxt.open(NameFile_txt, std::ofstream::out | std::ofstream::app);
		for (int i = 0; i < maxMinor; i++)
		{
			int j = i;
			//X
			int Nx = j / (NbPoints*NbPoints);
			float x = ((float)(Nx)*P_Simulation_DEVICE->X.step) + P_Simulation_DEVICE->X.start;
			// on retranche 
			j -= (Nx)*(NbPoints*NbPoints);

			//Y
			int Ny = j / (NbPoints);
			float y = ((float)(Ny)*P_Simulation_DEVICE->Y.step) + P_Simulation_DEVICE->Y.start;
			// on retranche 
			j -= (Ny)*(NbPoints);

			//Z
			int Nz = j;
			float z = (float)(Nz)*P_Simulation_DEVICE->Z.step + P_Simulation_DEVICE->Z.start;
			// on retranche Q2
			j -= Nz;

			//W
			int Nw = j;
			//printf("index = %d  - Z Tempindex = %d \n", i, Tempindex);
			float w = (((float)Nw)*P_Simulation_DEVICE->W.step) + P_Simulation_DEVICE->W.start;

			short iter = Tab_Iter[i];
			if (iter > 0)
			{
				if (FilterQ_H(&Filter,&Nx, &Ny, &Nz, &Nw, (int)iter, P_Simulation_DEVICE))
				{
					;
					file << x << ";" << y << ";" << z << ";" << w << ";" << iter << ";\n";
					if (x > Stat.Xmax)
						Stat.Xmax = x;
					if (x < Stat.Xmin)
						Stat.Xmin = x;


					if (y > Stat.Ymax)
						Stat.Ymax = y;
					if (y < Stat.Ymin)
						Stat.Ymin = y;

					if (z > Stat.Zmax)
						Stat.Zmax = z;
					if (z < Stat.Zmin)
						Stat.Zmin = z;

					if (w > Stat.Wmax)
						Stat.Wmax = w;
					if (w < Stat.Wmin)
						Stat.Wmin = w;

					Stat.NbPoint++;
					filetxt << x << ";" << y << ";" << z << "\n";
				}

			}

		}
		filetxt.close();
		file.close();
		if (IsShow)
			std::cout << "Write csv -->  End" << "\n";

		if (IsShow)
			std::cout << "Clear Mem + Reste  -->  Start" << "\n";
		hipFree(P_Simulation_DEVICE);
		hipFree(Tab_Iter);
		//hipDeviceReset();
		if (IsShow)
			std::cout << "Clear Mem + Reste  -->  End" << "\n";
	}

	file.open(NameFile_stat, std::ofstream::out | std::ofstream::app);
	file << "Statistiques : \n";
	file << "				X min = " << Stat.Xmin << "\n";
	file << "				X max = " << Stat.Xmax << "\n";
	file << "				Y min = " << Stat.Ymin << "\n";
	file << "				Y max = " << Stat.Ymax << "\n";
	file << "				Z min = " << Stat.Zmin << "\n";
	file << "				Z max = " << Stat.Zmax << "\n";
	file << "				W min = " << Stat.Wmin << "\n";
	file << "				W max = " << Stat.Wmax << "\n";
	file << "				NbPoint plot = " << Stat.NbPoint << "\n";
	file.close();
	return 0;
}
