#include "hip/hip_runtime.h"
#include "compute.cuh"

// CUDA kernel to Compute itermax of quaternion
__global__ void kernel(const struct_P_Simulation_T *P_Simulation, int *Tab_Iter)
{
	//int Tempindex = 0;
	struct_Q_T Q_Current;
	float w, x, y, z;
	int iter = 0;
	//X
	x = ((float)blockIdx.x)*P_Simulation->X.step + P_Simulation->X.start;
	//Y
	y = ((float)blockIdx.y)*P_Simulation->Y.step + P_Simulation->Y.start;
	//Z
	z = ((float)blockIdx.z)*P_Simulation->Z.step + P_Simulation->Z.start;
	//W
	w = ((float)threadIdx.x)*P_Simulation->W.step + P_Simulation->W.start;

	CreateQ_By_float(&Q_Current, x, y, z, w);

	for (iter = 0; iter <= P_Simulation->Iter.max; iter++)
	{
		Get_QPow(&Q_Current, P_Simulation->power);
		Q_Current.x += x;
		Q_Current.y += y;
		Q_Current.z += z;
		Q_Current.w += w;

		if (Get_QNorm(&Q_Current) > P_Simulation->rMax)
			goto Fin;
	}
Fin:
	if (iter > 0)
		iter--;
	int index = blockIdx.x*P_Simulation->X.coef + blockIdx.y*P_Simulation->Y.coef + blockIdx.z*P_Simulation->Z.coef + threadIdx.x*P_Simulation->W.coef;
	if (index < P_Simulation->max)
		Tab_Iter[index] = iter;// index % 255;
	else
		printf("%d > %d", index, P_Simulation->max);
}



struct_Stat_float_T compute(struct_P_All_T Config, bool state)
{
std::ofstream filetxt;
std::ofstream file;
	
int Tab_Histo[300];
int  Nbpoint_iter = 0;
int NoConfig = 0;
int NbConfig = Config.W.NbStep*Config.X.NbStep*Config.Y.NbStep*Config.Z.NbStep;
//Stat
struct_Stat_float_T Stat;
for (int NoW = 0; NoW < Config.W.NbStep; NoW++)
{
	for (int NoX = 0; NoX < Config.X.NbStep; NoX++)
	{
		for (int NoY = 0; NoY < Config.Y.NbStep; NoY++)
		{
			for (int NoZ = 0; NoZ < Config.Z.NbStep; NoZ++)
			{
				NoConfig++;
				std::cout << "---------------------------------------------------\n";
				std::cout << "Config  " << NoConfig << " sur " << NbConfig << "\n";

				float W = NoW*Config.W.step + Config.W.start;
				float X = NoX*Config.X.step + Config.X.start;
				float Y = NoY*Config.Y.step + Config.Y.start;
				float Z = NoZ*Config.Z.step + Config.Z.start;

				int PasW = Config.NbPointByStep;
				if (Config.W.NbStep == 1)
					PasW = 1;

				int PasX = Config.NbPointByStep;
				if (Config.X.NbStep == 1)
					PasX = 1;

				int PasY = Config.NbPointByStep;
				if (Config.Y.NbStep == 1)
					PasY = 1;

				int PasZ = Config.NbPointByStep;
				if (Config.Z.NbStep == 1)
					PasZ = 1;

				//Taille de tableau
				int max = PasZ*PasY*PasX*PasW;

				if (Config.isShow)
					std::cout << "hipMallocManaged Config  -->  Start" << "\n";

				struct_P_Simulation_T *P_Simulation;
				int *Tab_Iter;
				// Allocate Unified Memory -- accessible from CPU or GPU
				hipMallocManaged(&P_Simulation, sizeof(struct_P_Simulation_T));
				hipMallocManaged(&Tab_Iter, max * sizeof(int));
				if (Config.isShow)
					std::cout << "hipMallocManaged Config  -->  End " << "\n";

				if (Config.isShow)
					std::cout << "P_Simulation Config  -->  Start" << "\n";
				// Pramatrage de W
				P_Simulation->W.start = W;
				P_Simulation->W.end = W + Config.W.step;
				P_Simulation->W.NbStep = PasW;
				if (PasW == 1)
					P_Simulation->W.step = 0.0f;
				else
					P_Simulation->W.step = (Config.W.step) / (PasW - 1);
				P_Simulation->W.coef = PasX*PasY*PasZ;

				// Pramatrage de X
				P_Simulation->X.start = X;
				P_Simulation->X.end = X + Config.X.step;
				P_Simulation->X.NbStep = PasX;
				if (PasX == 1)
					P_Simulation->X.step = 0.0f;
				else
					P_Simulation->X.step = (Config.X.step) / (PasX - 1);
				P_Simulation->X.coef = PasY*PasZ;

				// Pramatrage de Y
				P_Simulation->Y.start = Y;
				P_Simulation->Y.end = Y + Config.Y.step;
				P_Simulation->Y.NbStep = PasY;
				if (PasY == 1)
					P_Simulation->Y.step = 0.0f;
				else
					P_Simulation->Y.step = (Config.Y.step) / (PasY - 1);
				P_Simulation->Y.coef = PasZ;

				// Pramatrage de Z
				P_Simulation->Z.start = Z;
				P_Simulation->Z.end = Z + Config.Z.step;
				P_Simulation->Z.NbStep = PasZ;
				if (PasZ == 1)
					P_Simulation->Z.step = 0.0f;
				else
					P_Simulation->Z.step = (Config.Z.step) / (PasZ - 1);
				P_Simulation->Z.coef = 1;

				//Stat Step

				Stat.Wstep = P_Simulation->W.step;
				Stat.Xstep = P_Simulation->X.step;
				Stat.Ystep = P_Simulation->Y.step;
				Stat.Zstep = P_Simulation->Z.step;


				//Parametrage Iter
				P_Simulation->Iter.max = Config.Iter.max;
				P_Simulation->Iter.min = Config.Iter.min;

				//Parametrage Power
				P_Simulation->power = Config.power;

				//Parametrage Rmax
				P_Simulation->rMax = Config.rMax;

				//Parametrage max
				P_Simulation->max = max;

				if (Config.isShow)
					std::cout << "P_Simulation Config  -->  End" << "\n";

				if (Config.isShow)
					std::cout << "Tab_Iter and Tab_Histo Init  -->  Start" << "\n";
				for (int i = 0; i < max; i++)
					Tab_Iter[i] = 0;

				for (int i = 0; i <= Config.Iter.max; i++)
					Tab_Histo[i] = 0;
				if (Config.isShow)
					std::cout << "Tab_Iter and Tab_Histo Init -->  End" << "\n";

				if (Config.isShow)
					std::cout << "Compude GPU -->  Start" << "\n";
				dim3 grid(PasX, PasY, PasZ);
				dim3 block(PasW, 1, 1);
				kernel << <grid, block >> >(P_Simulation, Tab_Iter);
				if (Config.isShow)
					std::cout << "Compude GPU -->  End" << "\n";


				if (Config.isShow)
					std::cout << "hipDeviceSynchronize-->  Start" << "\n";
				hipDeviceSynchronize();
				if (Config.isShow)
					std::cout << "hipDeviceSynchronize -->  End" << "\n";

				if (Config.isShow)
					std::cout << "Analyzer Simulation -->  Start" << "\n";
				Nbpoint_iter = 0;
				for (int i = 0; i < max; i++)
				{
					if (Tab_Iter[i] > 0)
						Nbpoint_iter++;
					Tab_Histo[Tab_Iter[i]]++;
				}
				if (Config.isShow)
				{
					std::cout << "Nb point Nbpoint_iter = " << Nbpoint_iter << "\n";
					std::cout << "Soit  :  " << (float)(Nbpoint_iter / ((float)max / 10000.0f)) / 100.0f << "%  soit " << Nbpoint_iter << "pt sur " << max << "pt \n";
					std::cout << "Analyzer Simulation -->  End" << "\n";
				}

				if (Config.isShow)
					std::cout << "Write Histogram -->  Start" << "\n";
				file.open(Config.nameFile.histo, std::ofstream::out | std::ofstream::app);
				file << NoConfig << ";";
				for (int i = 0; i <= Config.Iter.max; i++)
					file << Tab_Histo[i] << ";";
				file << "\n";
				file.close();
				if (Config.isShow)
					std::cout << "Write Histogram -->  End" << "\n";


				file.open(Config.nameFile.csv, std::ofstream::out | std::ofstream::app);
				filetxt.open(Config.nameFile.txt, std::ofstream::out | std::ofstream::app);


				for (int i = 0; i < max; i++)
				{
					int j = i;

					//W
					int iW = 0;
					if (PasW > 1)
					{
						iW = j / P_Simulation->W.coef;
					}
					//printf("index = %d  - Z Tempindex = %d \n", i, Tempindex);
					float w = (float)iW*P_Simulation->W.step + P_Simulation->W.start;
					// on retranche 
					j -= iW*P_Simulation->W.coef;

					//X
					int iX = 0;
					if (PasX > 1)
					{
						iX = j / P_Simulation->X.coef;
					}
					float x = (float)iX*P_Simulation->X.step + P_Simulation->X.start;
					// on retranche 
					j -= iX*P_Simulation->X.coef;

					//Y
					int iY = 0;
					if (PasY > 1)
					{
						iY = j / P_Simulation->Y.coef;
					}
					float y = (float)iY*P_Simulation->Y.step + P_Simulation->Y.start;
					// on retranche 
					j -= iY*P_Simulation->Y.coef;

					//Z
					int iZ = 0;
					if (PasZ > 1)
					{
						iZ = j / P_Simulation->Z.coef;
					}
					float z = (float)iZ*P_Simulation->Z.step + P_Simulation->Z.start;
					// on retranche 
					j -= iZ*P_Simulation->Z.coef;


					int iter = Tab_Iter[i];
					if ((iter >= Config.Iter.min && Config.Iter.isFix == 2) || (iter == Config.Iter.min && Config.Iter.isFix == 1))
					{
						int filter = Config.filter;
						if (state == true)
						{
							if (FilterQ_H(&filter, &iX, &iY, &iZ, &iW, iter, P_Simulation))
							{
								file << x << ";" << y << ";" << z << ";" << w << ";" << iter << ";\n";
								int NbDim = 0;
								if (PasW > 1)
								{
									filetxt << w << ";";
									NbDim++;
								}

								if (PasX > 1)
								{
									filetxt << x << ";";
									NbDim++;
								}

								if (PasY > 1)
								{
									filetxt << y << ";";
									NbDim++;
								}

								if (PasZ > 1)
								{
									filetxt << z << ";";
									NbDim++;
								}

								if (NbDim >= 3)
									filetxt << iter << "\n";
								else
									filetxt << ((float)iter) / ((float)Config.Iter.max)*3.0f << "\n";
							}
						}
						if (w > Stat.Wmax)
							Stat.Wmax = w;
						if (w < Stat.Wmin)
							Stat.Wmin = w;

						if (x > Stat.Xmax)
							Stat.Xmax = x;
						if (x < Stat.Xmin)
							Stat.Xmin = x;

						if (y > Stat.Ymax)
							Stat.Ymax = y;
						if (y < Stat.Ymin)
							Stat.Ymin = y;

						if (z > Stat.Zmax)
							Stat.Zmax = z;
						if (z < Stat.Zmin)
							Stat.Zmin = z;

						Stat.NbPoint++;
					}

				}
				file.close();
				filetxt.close();

				if (Config.isShow)
					std::cout << "Clear Mem + Reste  -->  Start" << "\n";
				hipFree(P_Simulation);
				hipFree(Tab_Iter);
				//hipDeviceReset();
				if (Config.isShow)
					std::cout << "Clear Mem + Reste  -->  End" << "\n";
			}
		}
	}
}

return Stat;

}