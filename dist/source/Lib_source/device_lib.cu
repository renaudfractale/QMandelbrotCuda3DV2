#include "hip/hip_runtime.h"
#include "device_lib.cuh"


__device__  void CreateQ_By_float(struct_Q_T *out, float x, float y, float z, float w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

__device__  float  Get_QNorm(struct_Q_T *Q)
{
	return sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
}

__device__ void Get_QPow(struct_Q_T *Q, float pow)
{
	float A = Get_QNorm(Q);
	float theta = 0.0f;
	float B = 0.0f;
	float R = 0.0f;
	if (pow > 0.0f && A>0.000001f)
	{
		float coef = 1.0f;
		if (A<1.0f)
		{
			//printf("%f *******\n", A);
			coef = 1 / A;
			Q->x *= coef;
			Q->y *= coef;
			Q->z *= coef;
			Q->w *= coef;

		}
		A = Get_QNorm(Q);
		//printf("%f +++++++++\n", A);
		theta = acosf(Q->w / A)*pow;
		B = sqrt(A*A - Q->w*Q->w);
		R = exp2f(logf(A / coef)* pow);
		Q->x = R*sinf(theta)*(Q->x / B);
		Q->y = R*sinf(theta)*(Q->y / B);
		Q->z = R*sinf(theta)*(Q->z / B);
		Q->w = R*cosf(theta);

	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0f;
		Q->x = 0.0f;
		Q->y = 0.0f;
		Q->z = 0.0f;

	}
}