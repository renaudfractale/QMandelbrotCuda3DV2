#include "hip/hip_runtime.h"
#include "host_lib.cuh"


__host__  void CreateQ_By_float_H(struct_Q_T *out, float x, float y, float z, float w)
{
	out->x = x;
	out->y = y;
	out->z = z;
	out->w = w;
}

__host__  float  Get_QNorm_H(struct_Q_T *Q)
{
	return sqrtf(Q->x*Q->x + Q->y*Q->y + Q->z*Q->z + Q->w*Q->w);
}

__host__ void Get_QPow_H(struct_Q_T *Q, float pow)
{
	float A = Get_QNorm_H(Q);
	float theta = 0.0f;
	float B = 0.0f;
	float R = 0.0f;
	if (pow > 0.0f && A>0.000001f)
	{
		float coef = 1.0f;
		if (A<1.0f)
		{
			//printf("%f *******\n", A);
			coef = 1 / A;
			Q->x *= coef;
			Q->y *= coef;
			Q->z *= coef;
			Q->w *= coef;

		}
		A = Get_QNorm_H(Q);
		//printf("%f +++++++++\n", A);
		theta = acosf(Q->w / A)*pow;
		B = sqrt(A*A - Q->w*Q->w);
		R = exp2f(logf(A / coef)* pow);
		Q->x = R*sinf(theta)*(Q->x / B);
		Q->y = R*sinf(theta)*(Q->y / B);
		Q->z = R*sinf(theta)*(Q->z / B);
		Q->w = R*cosf(theta);

	}
	else
	{
		//printf("%f --------\n", A);
		Q->w = 0.0f;
		Q->x = 0.0f;
		Q->y = 0.0f;
		Q->z = 0.0f;

	}
}

__host__ int  GetQIter_H(struct_P_Simulation_T *P_Simulation_DEVICE, int  *x_filter, int  *y_filter, int *z_filter, int *w_filter)
{
	//int Tempindex = 0;
	struct_Q_T Q_Current;
	float w, x, y, z;
	int iter_computed;
	//X
	x = ((float)*x_filter)*P_Simulation_DEVICE->X.step + P_Simulation_DEVICE->X.start;
	//Y
	y = ((float)*y_filter)*P_Simulation_DEVICE->Y.step + P_Simulation_DEVICE->Y.start;
	//Z
	z = ((float)*z_filter)*P_Simulation_DEVICE->Z.step + P_Simulation_DEVICE->Z.start;
	//W
	w = ((float)*w_filter)*P_Simulation_DEVICE->W.step + P_Simulation_DEVICE->W.start;



	CreateQ_By_float_H(&Q_Current, x, y, z, w);

	for (iter_computed = 0; iter_computed <= P_Simulation_DEVICE->Iter.max; iter_computed++)
	{
		Get_QPow_H(&Q_Current, P_Simulation_DEVICE->power);
		Q_Current.x += x;
		Q_Current.y += y;
		Q_Current.z += z;
		Q_Current.w += w;

		if (Get_QNorm_H(&Q_Current) > P_Simulation_DEVICE->rMax)
		{
			if (iter_computed > 0)
				iter_computed--;
			return iter_computed;
		}
	}
	if (iter_computed > 0)
		iter_computed--;
	return iter_computed;
}
__host__ bool  FilterQ_H(int *Filter, int *Nx, int *Ny, int *Nz, int *Nw, int iter,	struct_P_Simulation_T *P_Simulation)
{
	if (*Filter == 0)
		return true;
	int iter_computed = 0;

	int pasx = 1;
	if (P_Simulation->X.NbStep == 1)
		pasx = 0;

	int pasy = 1;
	if (P_Simulation->Y.NbStep == 1)
		pasy = 0;

	int pasz = 1;
	if (P_Simulation->Z.NbStep == 1)
		pasz = 0;

	int pasw = 1;
	if (P_Simulation->W.NbStep == 1)
		pasw = 0;


	for (int x_filter = *Nx - pasx; x_filter <= *Nx + pasx; x_filter++)
	{
		for (int y_filter = *Ny - pasy; y_filter <= *Ny + pasy; y_filter++)
		{
			for (int z_filter = *Nz - pasz; z_filter <= *Nz + pasz; z_filter++)
			{
				for (int w_filter = *Nw - pasw; w_filter <= *Nw + pasw; w_filter++)
				{
					iter_computed = GetQIter_H(P_Simulation, &x_filter, &y_filter, &z_filter, &w_filter);
					if (*Filter == 1)
					{
						if (iter_computed != iter)
							return true;
					}
					else //filter==2
					{
						if (iter_computed == 0)
							return true;
					}
				}
			}
		}
	}
	return false;
}
