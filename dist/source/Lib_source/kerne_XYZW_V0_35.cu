#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //lib W10
#include ""//lib W10
#include "compute.cuh"


int main(int argc, char *argv[])
{
	struct_P_All_T Config;
    int state= menu( argc, argv, &Config);

	if (state != 0)
		return state;

	/********  Clear File ************/
	std::ofstream filetxt;
	filetxt.open(Config.nameFile.txt);
	filetxt.close();

	std::ofstream file;
	file.open(Config.nameFile.csv);
	file << "X;Y;Z;W;iter;\n";
	file.close();


	file.open(Config.nameFile.histo);
	file << "index;";
	for (int i = 0; i <= Config.Iter.max; i++)
		file << i << ";";
	file << "\n";
	file.close();

	//Affiche Stat
	FILE *fileman;
	char line[1000];
	fileman = fopen(Config.nameFile.stat, "r");
	if (fileman != NULL)
	{
		while (std::fgets(line, 1000, fileman))
		{
			std::cout << line;
		}
		fclose(fileman);
	}
	else
		std::cout << "Error  : "<< Config.nameFile.stat <<" no found" << "\n";
	hipSetDevice(Config.dev);


	struct_Stat_float_T Stat;

	if(Config.modeAM==false)
		Stat =compute(Config, true);
	else
	{
		bool Dim[] = {false,false ,false ,false };
		
		if (Config.W.isFix == 2)
			Dim[0] = true;
		if (Config.X.isFix == 2)
			Dim[1] = true;
		if (Config.Y.isFix == 2)
			Dim[2] = true;
		if (Config.Z.isFix == 2)
			Dim[3] = true;

		int NbDim = 0;

		for (int i = 0; i <= 3; i++)
		{
			if (Dim[i]==true)
			{
				NbDim++;
				switch (i)
				{
				case 0:
					Config.W.isFix = 2;
					Config.W.start = -20.0f;
					Config.W.end = 20.0f;
					Config.W.NbStep = 2;
					Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
					break;
				case 1:
					Config.X.isFix = 2;
					Config.X.start = -20.0f;
					Config.X.end = 20.0f;
					Config.X.NbStep = 2;
					Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
					break;
				case 2:
					Config.Y.isFix = 2;
					Config.Y.start = -20.0f;
					Config.Y.end = 20.0f;
					Config.Y.NbStep = 2;
					Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
					break;
				case 3:
					Config.Z.isFix = 2;
					Config.Z.start = -20.0f;
					Config.Z.end = 20.0f;
					Config.Z.NbStep = 2;
					Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
					break;
				default:
					break;
				}
			}	
		}
		//si et selement si il y a 3 dimentions
		if (NbDim == 3)
		{
			float pas = 10;
			float OldPas = 20;
			int nb_iter = 10;
			for (int iter = 0; iter <= nb_iter; iter++)
			{
				Stat = compute(Config,false);
				// r�tro action
				for (int i = 0; i <= 3; i++)
				{
					if (Dim[i] == true)
					{
						switch (i)
						{
						case 0:
							if (Stat.Wmin == Config.W.start)
							{
								Config.W.start -= OldPas;
								if (nb_iter != i)
									Config.W.start += pas;
							}
							else if(nb_iter!=i)
								Config.W.start +=pas;

							if (Stat.Wmax == Config.W.end)
							{
								Config.W.end += OldPas;
								if (nb_iter != i)
									Config.W.end -= pas;
							}								
							else if(nb_iter != i)
								Config.W.end -= pas;
							Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
							break;
						case 1:
							if (Stat.Xmin == Config.X.start)
							{
								Config.X.start -= OldPas;
								if (nb_iter != i)
									Config.X.start += pas;
							}
							else if (nb_iter != i)
								Config.X.start += pas;
							if (Stat.Xmax == Config.X.end)
							{
								Config.X.end += OldPas;
								if (nb_iter != i)
									Config.X.end -= pas;
							}
							else if (nb_iter != i)
								Config.X.end -= pas;
							Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
							break;
						case 2:
							if (Stat.Ymin == Config.Y.start)
							{
								Config.Y.start -= OldPas;
								if (nb_iter != i)
									Config.Y.start += pas;
							}
							else if (nb_iter != i)
								Config.Y.start += pas;
							if (Stat.Ymax == Config.Y.end)
							{
								Config.Y.end += OldPas;
								if (nb_iter != i)
									Config.Y.end -= pas;
							}
							else if (nb_iter != i)
								Config.Y.end -= pas;
							Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
							break;
						case 3:
							if (Stat.Zmin == Config.Z.start)
							{
								Config.Z.start -= OldPas;
								if (nb_iter != i)
									Config.Z.start += pas;
							}
							else if (nb_iter != i)
								Config.Z.start += pas;
							if (Stat.Zmax == Config.Z.end)
							{
								Config.Z.end += OldPas;
								if (nb_iter != i)
									Config.Z.end -= pas;
							}
							else if (nb_iter != i)
								Config.Z.end -= pas;
							Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
							break;
						default:
							break;
						}
					}

				}

				file.open(Config.nameFile.stat, std::ofstream::out | std::ofstream::app);
				file << "Statistiques : \n";
				file << "				X min = " << Stat.Xmin << "\n";
				file << "				X max = " << Stat.Xmax << "\n";
				file << "				Y min = " << Stat.Ymin << "\n";
				file << "				Y max = " << Stat.Ymax << "\n";
				file << "				Z min = " << Stat.Zmin << "\n";
				file << "				Z max = " << Stat.Zmax << "\n";
				file << "				W min = " << Stat.Wmin << "\n";
				file << "				W max = " << Stat.Wmax << "\n";
				file << "				X step = " << Stat.Xstep << "\n";
				file << "				Y step = " << Stat.Ystep << "\n";
				file << "				Z step = " << Stat.Zstep << "\n";
				file << "				W step = " << Stat.Wstep << "\n";
				file << "				NbPoint plot = " << Stat.NbPoint << "\n";
				file.close();


				OldPas = pas;
				pas = pas / 2.0f;
			}

			for (int i = 0; i <= 3; i++)
			{
				if (Dim[i] == true)
				{
					switch (i)
					{
					case 0:
						Config.W.NbStep = 5;
						Config.W.step = (Config.W.end - Config.W.start) / ((float)Config.W.NbStep);
						break;
					case 1:
						Config.X.NbStep = 5;
						Config.X.step = (Config.X.end - Config.X.start) / ((float)Config.X.NbStep);
						break;
					case 2:
						Config.Y.NbStep = 5;
						Config.Y.step = (Config.Y.end - Config.Y.start) / ((float)Config.Y.NbStep);
						break;
					case 3:
						Config.Z.NbStep = 5;
						Config.Z.step = (Config.Z.end - Config.Z.start) / ((float)Config.Z.NbStep);
						break;
					default:
						break;
					}
				}

			}
			if(Stat.NbPoint!=0)
				Stat = compute(Config, true);
		}
		else
		{
			std::cout << "Error  mode: mode auto only 3 dim\n";
			return -1;
		}

	}
	
	file.open(Config.nameFile.stat, std::ofstream::out | std::ofstream::app);
	file << "Statistiques : \n";
	file << "				X min = " << Stat.Xmin << "\n";
	file << "				X max = " << Stat.Xmax << "\n";
	file << "				Y min = " << Stat.Ymin << "\n";
	file << "				Y max = " << Stat.Ymax << "\n";
	file << "				Z min = " << Stat.Zmin << "\n";
	file << "				Z max = " << Stat.Zmax << "\n";
	file << "				W min = " << Stat.Wmin << "\n";
	file << "				W max = " << Stat.Wmax << "\n";
	file << "				X step = " << Stat.Xstep << "\n";
	file << "				Y step = " << Stat.Ystep << "\n";
	file << "				Z step = " << Stat.Zstep << "\n";
	file << "				W step = " << Stat.Wstep << "\n";
	file << "				NbPoint plot = " << Stat.NbPoint << "\n";
	file.close();




	return 0;//Fin du programme
}